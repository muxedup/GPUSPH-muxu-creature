#include "hip/hip_runtime.h"
#ifndef PROBLEM_BC_CU
#define PROBLEM_BC_CU

#include <math.h>
#include <string>
#include <iostream>

#include "LaPalisse.h"
#include "GlobalData.h"
#include "textures.cuh"
#include "utils.h"
#include "Problem.h"

namespace cuLaPalisse
{
#include "cuda/cellgrid.cuh"
// Core SPH functions
#include "cuda/sph_core_utils.cuh"

__device__
void
LaPalisse_imposeBoundaryCondition(
	const	particleinfo	info,
	const	float3			absPos,
			float			waterdepth,
	const	float			t,
			float4&			vel,
			float4&			eulerVel,
			float&			tke,
			float&			eps)
{
	vel = make_float4(0.0f);
	tke = 0.0f;
	eps = 0.0f;

	// open boundary conditions
	if (IO_BOUNDARY(info)) {
		// impose pressure
		if (!VEL_IO(info)) {
			if (object(info)==1) {
				// set inflow waterdepth
				waterdepth = INLET_WATER_LEVEL - 1.08f;
			}
			const float localdepth = fmax(waterdepth - absPos.z, 0.0f);
			const float pressure = 9.81e3f*localdepth;
			eulerVel.w = RHO(pressure, fluid_num(info));
		}
	}
}

__global__ void
LaPalisse_imposeBoundaryConditionDevice(
			float4*		newVel,
			float4*		newEulerVel,
			float*		newTke,
			float*		newEpsilon,
	const	float4*		oldPos,
	const	uint*		IOwaterdepth,
	const	float		t,
	const	uint		numParticles,
	const	hashKey*	particleHash)
{
	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;

	if (index >= numParticles)
		return;

	float4 vel = make_float4(0.0f);			// imposed velocity for moving objects
	float4 eulerVel = make_float4(0.0f);	// imposed velocity/pressure for open boundaries
	float tke = 0.0f;						// imposed turbulent kinetic energy for open boundaries
	float eps = 0.0f;						// imposed turb. diffusivity for open boundaries

	if(index < numParticles) {
		const particleinfo info = tex1Dfetch(infoTex, index);
		// open boundaries and forced moving objects
		// the case of a corner needs to be treated as follows:
		// - for a velocity inlet nothing is imposed (in case of k-eps newEulerVel already contains the info
		//   from the viscosity
		// - for a pressure inlet the pressure is imposed on the corners. If we are in the k-epsilon case then
		//   we need to get the viscosity info from newEulerVel (x,y,z) and add the imposed density in .w
		if (VERTEX(info) && IO_BOUNDARY(info) && (!CORNER(info) || !VEL_IO(info))) {
			// For corners we need to get eulerVel in case of k-eps and pressure outlet
			if (CORNER(info) && newTke && !VEL_IO(info))
				eulerVel = newEulerVel[index];
			const float3 absPos = d_worldOrigin + as_float3(oldPos[index])
									+ calcGridPosFromParticleHash(particleHash[index])*d_cellSize
									+ 0.5f*d_cellSize;
			float waterdepth = 0.0f;
			if (!VEL_IO(info) && IOwaterdepth) {
				waterdepth = ((float)IOwaterdepth[object(info)-1])/((float)UINT_MAX); // now between 0 and 1
				waterdepth *= d_cellSize.z*d_gridSize.z; // now between 0 and world size
				waterdepth += d_worldOrigin.z; // now absolute z position
			}
			// this now calls the virtual function that is problem specific
			LaPalisse_imposeBoundaryCondition(info, absPos, waterdepth, t, vel, eulerVel, tke, eps);
			// copy values to arrays
			newVel[index] = vel;
			newEulerVel[index] = eulerVel;
			if(newTke)
				newTke[index] = tke;
			if(newEpsilon)
				newEpsilon[index] = eps;
		}
	}
}

} // end of cuLaPalisse namespace

extern "C"
{

void
LaPalisse::setboundconstants(
	const	PhysParams	*physparams,
	float3	const&		worldOrigin,
	uint3	const&		gridSize,
	float3	const&		cellSize)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuLaPalisse::d_worldOrigin), &worldOrigin, sizeof(float3)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuLaPalisse::d_cellSize), &cellSize, sizeof(float3)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuLaPalisse::d_gridSize), &gridSize, sizeof(uint3)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuLaPalisse::d_rho0), &physparams->rho0, MAX_FLUID_TYPES*sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuLaPalisse::d_bcoeff), &physparams->bcoeff, MAX_FLUID_TYPES*sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuLaPalisse::d_gammacoeff), &physparams->gammacoeff, MAX_FLUID_TYPES*sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuLaPalisse::d_sscoeff), &physparams->sscoeff, MAX_FLUID_TYPES*sizeof(float)));

}

}

void
LaPalisse::imposeBoundaryConditionHost(
			float4*			newVel,
			float4*			newEulerVel,
			float*			newTke,
			float*			newEpsilon,
	const	particleinfo*	info,
	const	float4*			oldPos,
			uint			*IOwaterdepth,
	const	float			t,
	const	uint			numParticles,
	const	uint			numObjects,
	const	uint			particleRangeEnd,
	const	hashKey*		particleHash)
{
	uint numThreads = min(BLOCK_SIZE_IOBOUND, particleRangeEnd);
	uint numBlocks = div_up(particleRangeEnd, numThreads);

	int dummy_shared = 0;
	// TODO: Probably this optimization doesn't work with this function. Need to be tested.
	#if (__COMPUTE__ == 20)
	dummy_shared = 2560;
	#endif

	CUDA_SAFE_CALL(hipBindTexture(0, infoTex, info, numParticles*sizeof(particleinfo)));

	cuLaPalisse::LaPalisse_imposeBoundaryConditionDevice<<< numBlocks, numThreads, dummy_shared >>>
		(newVel, newEulerVel, newTke, newEpsilon, oldPos, IOwaterdepth, t, numParticles, particleHash);

	CUDA_SAFE_CALL(hipUnbindTexture(infoTex));

	// reset waterdepth calculation
	if (IOwaterdepth) {
		uint h_IOwaterdepth[numObjects];
		for (uint i=0; i<numObjects; i++)
			h_IOwaterdepth[i] = 0;
		CUDA_SAFE_CALL(hipMemcpy(IOwaterdepth, h_IOwaterdepth, numObjects*sizeof(int), hipMemcpyHostToDevice));
	}

	// check if kernel invocation generated an error
	CUT_CHECK_ERROR("imposeBoundaryCondition kernel execution failed");
}

#endif
