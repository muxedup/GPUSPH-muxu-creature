#include "hip/hip_runtime.h"
/*  Copyright 2011-2013 Alexis Herault, Giuseppe Bilotta, Robert A. Dalrymple, Eugenio Rustico, Ciro Del Negro

    Istituto Nazionale di Geofisica e Vulcanologia
        Sezione di Catania, Catania, Italy

    Università di Catania, Catania, Italy

    Johns Hopkins University, Baltimore, MD

    This file is part of GPUSPH.

    GPUSPH is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUSPH is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUSPH.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <cmath>
#include <iostream>

#include "Spheric2LJ.h"
#include "Cube.h"
#include "Point.h"
#include "Vector.h"
#include "GlobalData.h"
#include "cudasimframework.cu"

#define CENTER_DOMAIN 1
// set to coords (x,y,z) if more accuracy is needed in such point
// (waiting for relative coordinates)
#if CENTER_DOMAIN
#define OFFSET_X (-lx/2)
#define OFFSET_Y (-ly/2)
#define OFFSET_Z (-lz/2)
#else
#define OFFSET_X 0
#define OFFSET_Y 0
#define OFFSET_Z 0
#endif

Spheric2LJ::Spheric2LJ(GlobalData *_gdata) : Problem(_gdata)
{
	// Size and origin of the simulation domain
	lx = 3.22;
	ly = 1.0;
	lz = 1.0;
	H = 0.55;
	wet = false;
	m_usePlanes = get_option("use-planes", true);

	m_size = make_double3(lx, ly, lz);
	m_origin = make_double3(OFFSET_X, OFFSET_Y, OFFSET_Z);

	SETUP_FRAMEWORK(
		kernel<WENDLAND>,
		viscosity<ARTVISC>,
		//viscosity<SPSVISC>,
		//viscosity<DYNAMICVISC>,
		boundary<LJ_BOUNDARY>,
		add_flags<ENABLE_FERRARI>
	).select_options(
		m_usePlanes, add_flags<ENABLE_PLANES>()
	);

	// SPH parameters
	// ratio h / deltap (needs to be defined before calling set_deltap)
	simparams()->sfactor = 1.3;
	// set deltap (automatically computes h based on sfactor * deltap)
	set_deltap(0.02); //0.008
	simparams()->dtadaptfactor = 0.3;
	simparams()->buildneibsfreq = 10;
	simparams()->ferrari = 0.1;
	simparams()->tend = 1.0f;

	// Free surface detection
	addPostProcess(SURFACE_DETECTION);

	// Test points
	addPostProcess(TESTPOINTS);

	// Physical parameters
	physparams()->gravity = make_float3(0.0, 0.0, -9.81f);
	float g = length(physparams()->gravity);

	add_fluid(1000.0);
	set_equation_of_state(0,  7.0f, 20.f);

    //set p1coeff,p2coeff, epsxsph here if different from 12.,6., 0.5
	physparams()->dcoeff = 5.0f*g*H;
	physparams()->r0 = m_deltap;

	// BC when using MK boundary condition: Coupled with m_simsparams->boundarytype=MK_BOUNDARY
	#define MK_par 2
	physparams()->MK_K = g*H;
	physparams()->MK_d = 1.1*m_deltap/MK_par;
	physparams()->MK_beta = MK_par;
	#undef MK_par

	set_kinematic_visc(0, 1.0e-2f);
	physparams()->artvisccoeff = 0.3f;
	physparams()->epsartvisc = 0.01*simparams()->slength*simparams()->slength;

	// Drawing and saving times
	add_writer(VTKWRITER, 0.05);

	// Name of problem used for directory creation
	m_name = "Spheric2LJ";
}


Spheric2LJ::~Spheric2LJ(void)
{
	release_memory();
}


void Spheric2LJ::release_memory(void)
{
	parts.clear();
	obstacle_parts.clear();
	boundary_parts.clear();
}


int Spheric2LJ::fill_parts()
{
	float r0 = physparams()->r0;

	Cube fluid, fluid1;

	experiment_box = Cube(Point(m_origin), lx, ly, lz);

	obstacle = Cube(Point(m_origin + make_double3(2.3955, 0.295, 0.0)), 0.161, 0.403, 0.161);


	fluid = Cube(Point(m_origin + r0), 0.4, ly - 2*r0, H - r0);

	if (wet) {
		fluid1 = Cube(Point(m_origin + r0 + make_double3(H + m_deltap, 0, 0)),
			lx - H - m_deltap - 2*r0, 0.67 - 2*r0, 0.1);
	}

	boundary_parts.reserve(2000);
	parts.reserve(14000);

	if (!m_usePlanes) {
		experiment_box.SetPartMass(r0, physparams()->rho0[0]);
		experiment_box.FillBorder(boundary_parts, r0, false);
	}

	obstacle.SetPartMass(r0, physparams()->rho0[0]);
	obstacle.FillBorder(obstacle_parts, r0, true);

	fluid.SetPartMass(m_deltap, physparams()->rho0[0]);
	fluid.Fill(parts, m_deltap, true);
	if (wet) {
		fluid1.SetPartMass(m_deltap, physparams()->rho0[0]);
		fluid1.Fill(parts, m_deltap, true);
		obstacle.Unfill(parts, r0);
	}

	// Setting probes for Spheric2 test case
	//*******************************************************************
	// Wave gages
	add_gage(m_origin + make_double3(2.724, 0.5, 0.0));
	add_gage(m_origin + make_double3(2.228, 0.5, 0.0));
	add_gage(m_origin + make_double3(1.732, 0.5, 0.0));
	add_gage(m_origin + make_double3(0.582, 0.5, 0.0));
	// Pressure probes
	if (m_simframework->hasPostProcessEngine(TESTPOINTS)) {
		test_points.push_back(m_origin + make_double3(2.3955, 0.529, 0.021));
		test_points.push_back(m_origin + make_double3(2.3955, 0.529, 0.061));
		test_points.push_back(m_origin + make_double3(2.3955, 0.529, 0.101));
		test_points.push_back(m_origin + make_double3(2.3955, 0.529, 0.141));
		test_points.push_back(m_origin + make_double3(2.4165, 0.471, 0.161));
		test_points.push_back(m_origin + make_double3(2.4565, 0.471, 0.161));
		test_points.push_back(m_origin + make_double3(2.4965, 0.471, 0.161));
		test_points.push_back(m_origin + make_double3(2.5365, 0.471, 0.161));
	}
	//*******************************************************************

	return parts.size() + boundary_parts.size() + obstacle_parts.size() + test_points.size();
}

void Spheric2LJ::copy_planes(PlaneList& planes)
{
	if (!m_usePlanes) return;

	// bottom
	planes.push_back( implicit_plane(0, 0, 1.0, -m_origin.z) );
	// back
	planes.push_back( implicit_plane(1.0, 0, 0, -m_origin.x) );
	// front
	planes.push_back( implicit_plane(-1.0, 0, 0, m_origin.x + lx) );
	// side with smaller Y ("left")
	planes.push_back( implicit_plane(0, 1.0, 0, -m_origin.y) );
	// side with greater Y ("right")
	planes.push_back( implicit_plane(0, -1.0, 0, m_origin.y + ly) );
}

void Spheric2LJ::fillDeviceMap()
{
	// TODO: test which split performs better, if Y (not many particles passing) or X (smaller section)
	fillDeviceMapByAxis(Y_AXIS);
	//fillDeviceMapByEquation();
}

void Spheric2LJ::copy_to_array(BufferList &buffers)
{
	float4 *pos = buffers.getData<BUFFER_POS>();
	hashKey *hash = buffers.getData<BUFFER_HASH>();
	float4 *vel = buffers.getData<BUFFER_VEL>();
	particleinfo *info = buffers.getData<BUFFER_INFO>();

	for (uint i = 0; i < boundary_parts.size(); i++) {
		vel[i] = make_float4(0, 0, 0, physparams()->rho0[0]);
		info[i]= make_particleinfo(PT_BOUNDARY,0,i);
		calc_localpos_and_hash(boundary_parts[i], info[i], pos[i], hash[i]);
	}
	uint j = boundary_parts.size();
	if (boundary_parts.size() > 0)
		std::cout << "Boundary part mass:" << pos[j-1].w << "\n";
	else
		std::cout << "No boundary parts" << std::endl;

	//Testpoints
	if (test_points.size()) {
		std::cout << "\nTest points: " << test_points.size() << "\n";
		for (uint i = 0; i < test_points.size(); i++) {
			vel[i] = make_float4(0, 0, 0, physparams()->rho0[0]);
			info[i]= make_particleinfo(PT_TESTPOINT, 0, i);
			calc_localpos_and_hash(test_points[i], info[i], pos[i], hash[i]);
		}
		j += test_points.size();
		std::cout << "Test point mass:" << pos[j-1].w << "\n";
	}
	else
		std::cout << "No test points" << std::endl;

	std::cout << "Obstacle parts: " << obstacle_parts.size() << "\n";
	for (uint i = j; i < j + obstacle_parts.size(); i++) {
		vel[i] = make_float4(0, 0, 0, physparams()->rho0[0]);
		info[i]= make_particleinfo(PT_BOUNDARY,1,i);
		calc_localpos_and_hash(obstacle_parts[i-j], info[i], pos[i], hash[i]);
	}
	j += obstacle_parts.size();
	if (obstacle_parts.size() > 0)
		std::cout << "Obstacle part mass:" << pos[j-1].w << "\n";
	else
		std::cout << "No obstacle parts" << std::endl;

	std::cout << "Fluid parts: " << parts.size() << "\n";
	for (uint i = j; i < j + parts.size(); i++) {
		vel[i] = make_float4(0, 0, 0, physparams()->rho0[0]);
		info[i]= make_particleinfo(PT_FLUID,0,i);
		calc_localpos_and_hash(parts[i-j], info[i], pos[i], hash[i]);
	}
	j += parts.size();
	if (parts.size() > 0)
		std::cout << "Fluid part mass:" << pos[j-1].w << "\n";
	else
		std::cout << "No fluid parts" << std::endl;

	std::flush(std::cout);
}
