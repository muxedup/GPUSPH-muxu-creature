#include "hip/hip_runtime.h"
#include "XBuoyancyTest.h"
#include <cmath>
#include <iostream>

#include "GlobalData.h"
#include "cudasimframework.cu"
#include "Cube.h"
#include "Sphere.h"
#include "Point.h"
#include "Vector.h"


XBuoyancyTest::XBuoyancyTest(GlobalData *_gdata) : XProblem(_gdata)
{
	// Size and origin of the simulation domain
	double lx = 1.0;
	double ly = 1.0;
	double lz = 1.0;
	double H = 0.7;

	//m_size = make_double3(lx, ly, lz);
	//m_origin = make_double3(0.0, 0.0, 0.0);

	SETUP_FRAMEWORK(
		kernel<WENDLAND>,
		viscosity<ARTVISC>,
		//viscosity<SPSVISC>,
		//viscosity<KINEMATICVISC>,
		boundary<DYN_BOUNDARY>
	);

	// let the number of layers be autocomputed
	//setDynamicBoundariesLayers(4);

	// SPH parameters
	set_deltap(0.02); //0.008
	simparams()->dt = 0.0003f;
	simparams()->dtadaptfactor = 0.3;
	simparams()->buildneibsfreq = 10;
	simparams()->tend = 20.0f; //0.00036f

	// Physical parameters
	H = 0.6f;
	physparams()->gravity = make_float3(0.0, 0.0, -9.81f);
	double g = length(physparams()->gravity);
	add_fluid(1000.0);
	set_equation_of_state(0,  7.0f, 20.f);

	//set p1coeff,p2coeff, epsxsph here if different from 12.,6., 0.5
	physparams()->dcoeff = 5.0f*g*H;
	physparams()->r0 = m_deltap;

	set_kinematic_visc(0, 1.0e-6f);
	physparams()->artvisccoeff = 0.3f;
	physparams()->epsartvisc = 0.01*simparams()->slength*simparams()->slength;

	add_writer(VTKWRITER, 0.01);

	setPositioning(PP_CORNER);

	const int layers = 4;

	GeometryID cube = addBox(GT_FIXED_BOUNDARY, FT_BORDER, Point(0,0,0), lx, ly, lz);
	disableCollisions(cube);

	const double offs = m_deltap * layers;
	//addExtraWorldMargin(2*offs);

	GeometryID fluid = addBox(GT_FLUID, FT_SOLID, Point(offs, offs, offs),
		lx - 2.0 * offs, ly - 2.0 * offs, H);

	// TODO
	/*
	switch (object_type) {
		case 0: {
			olx, oly, olz = 10.0*m_deltap;
			cube  = Cube(Point(lx/2.0 - olx/2.0, ly/2.0 - oly/2.0, H/2.0 - olz/2.0), olx, oly, olz);
		case 1: {
			double r = 6.0*m_deltap;
			sphere = Sphere(Point(lx/2.0, ly/2.0, H/2.0 - r/4.0), r);
		case 2: // TORUS
	*/
	double R = lx * 0.2;
	double r = 4.0 * m_deltap;
	GeometryID torus = addTorus(GT_FLOATING_BODY, FT_BORDER, Point(lx/2.0, ly/2.0, H/2.0), R, r);
	setMassByDensity(torus, physparams()->rho0[0]*0.5);

	// Name of problem used for directory creation
	m_name = "XBuoyancyTest";
}

void XBuoyancyTest::ODE_near_callback(void *data, dGeomID o1, dGeomID o2)
{
	const int N = 10;
	dContact contact[N];

	int n = dCollide(o1, o2, N, &contact[0].geom, sizeof(dContact));
	for (int i = 0; i < n; i++) {
		contact[i].surface.mode = dContactBounce;
		contact[i].surface.mu   = dInfinity;
		contact[i].surface.bounce     = 0.0; // (0.0~1.0) restitution parameter
		contact[i].surface.bounce_vel = 0.0; // minimum incoming velocity for bounce
		dJointID c = dJointCreateContact(m_ODEWorld, m_ODEJointGroup, &contact[i]);
		dJointAttach (c, dGeomGetBody(contact[i].geom.g1), dGeomGetBody(contact[i].geom.g2));
	}
}
