#include "hip/hip_runtime.h"
/*
 * BuyancyTest.cc
 *
 *  Created on: 20 juin 2014
 *      Author: alexisherault
 */

#include "BuoyancyTest.h"
#include <cmath>
#include <iostream>

#include "GlobalData.h"
#include "cudasimframework.cu"
#include "Cube.h"
#include "Sphere.h"
#include "Point.h"
#include "Vector.h"


BuoyancyTest::BuoyancyTest(GlobalData *_gdata) : Problem(_gdata)
{
	// Size and origin of the simulation domain
	lx = 1.0;
	ly = 1.0;
	lz = 1.0;
	H = 0.7;

	m_size = make_double3(lx, ly, lz);
	m_origin = make_double3(0.0, 0.0, 0.0);

	SETUP_FRAMEWORK(
		kernel<WENDLAND>,
		viscosity<ARTVISC>,
		//viscosity<SPSVISC>,
		//viscosity<KINEMATICVISC>,
		boundary<DYN_BOUNDARY>
	);

	//addFilter(MLS_FILTER, 17);

	// SPH parameters
	set_deltap(0.02); //0.008
	simparams()->dt = 0.0003f;
	simparams()->dtadaptfactor = 0.3;
	simparams()->buildneibsfreq = 10;
	simparams()->tend = 5.0f; //0.00036f

	// Physical parameters
	H = 0.6f;
	physparams()->gravity = make_float3(0.0, 0.0, -9.81f);
	double g = length(physparams()->gravity);
	add_fluid(1000.0);
	set_equation_of_state(0,  7.0f, 20.f);

    //set p1coeff,p2coeff, epsxsph here if different from 12.,6., 0.5
	physparams()->dcoeff = 5.0f*g*H;
	physparams()->r0 = m_deltap;

	set_kinematic_visc(0, 1.0e-6f);
	physparams()->artvisccoeff = 0.3f;
	physparams()->epsartvisc = 0.01*simparams()->slength*simparams()->slength;
	physparams()->smagfactor = 0.12*0.12*m_deltap*m_deltap;
	physparams()->kspsfactor = (2.0/3.0)*0.0066*m_deltap*m_deltap;

	// Initialize ODE
	dInitODE();
	m_ODEWorld = dWorldCreate();
	m_ODESpace = dHashSpaceCreate(0);
	m_ODEJointGroup = dJointGroupCreate(0);
	dWorldSetGravity(m_ODEWorld, physparams()->gravity.x, physparams()->gravity.y, physparams()->gravity.z);	// Set gravity (x, y, z)

	//add_writer(VTKWRITER, 0.005);
	add_writer(VTKWRITER, 0.1);
	add_writer(COMMONWRITER, 0.0);

	// Name of problem used for directory creation
	m_name = "BuoyancyTest";
}


BuoyancyTest::~BuoyancyTest(void)
{
	release_memory();
}


void BuoyancyTest::release_memory(void)
{
	parts.clear();
	boundary_parts.clear();
}


int BuoyancyTest::fill_parts()
{
	const double dp = m_deltap;
	const int layers = 4;

	Cube experiment_box = Cube(Point(0, 0, 0), lx, ly, lz);

	Cube fluid = Cube(Point(dp*layers, dp*layers, dp*layers),
		lx - 2.0*dp*layers, ly - 2.0*dp*layers, H);
	planes[0] = dCreatePlane(m_ODESpace, 0.0, 0.0, 1.0, 0.0);
	planes[1] = dCreatePlane(m_ODESpace, 1.0, 0.0, 0.0, 0.0);
	planes[2] = dCreatePlane(m_ODESpace, -1.0, 0.0, 0.0, -lx);
	planes[3] = dCreatePlane(m_ODESpace, 0.0, 1.0, 0.0, 0.0);
	planes[4] = dCreatePlane(m_ODESpace, 0.0, -1.0, 0.0, -ly);

	boundary_parts.reserve(2000);
	parts.reserve(14000);

	experiment_box.SetPartMass(m_deltap, physparams()->rho0[0]);
	experiment_box.FillIn(boundary_parts, m_deltap, layers, false);
	fluid.SetPartMass(m_deltap, physparams()->rho0[0]);
	fluid.Fill(parts, m_deltap, true);

	const int object_type = 0;
	Object *floating;
	switch (object_type) {
		case 0: {
			double olx = 10.0*m_deltap;
			double oly = 10.0*m_deltap;
			double olz = 10.0*m_deltap;
			cube  = Cube(Point(lx/2.0 - olx/2.0, ly/2.0 - oly/2.0, H/2.0 - olz/2.0), olx, oly, olz);
			floating = &cube;
			}
			break;

		case 1: {
			double r = 6.0*m_deltap;
			sphere = Sphere(Point(lx/2.0, ly/2.0, H/2.0 - r/4.0), r);
			floating = &sphere;
			}
			break;

		case 2: {
			double R = lx*0.2;
			double r = 4.0*m_deltap;
			torus = Torus(Point(lx/2.0, ly/2.0, H/2.0), Vector(0, 0, 1), R, r);
			floating = &torus;
			}
			break;
	}

	floating->SetMass(m_deltap, physparams()->rho0[0]*0.5);
	floating->SetPartMass(m_deltap, physparams()->rho0[0]);
	floating->FillIn(floating->GetParts(), m_deltap, layers);
	floating->Unfill(parts, m_deltap*0.85);

	floating->ODEBodyCreate(m_ODEWorld, m_deltap);
	if (object_type != 2)
		floating->ODEGeomCreate(m_ODESpace, m_deltap);
	dBodySetLinearVel(floating->ODEGetBody(), 0.0, 0.0, 0.0);
	dBodySetAngularVel(floating->ODEGetBody(), 0.0, 0.0, 0.0);
	add_moving_body(floating, MB_ODE);
	floating->ODEPrintInformation();

	PointVect & rbparts = get_mbdata(uint(0))->object->GetParts();
	std::cout << "Rigid body " << 1 << ": " << rbparts.size() << " particles \n";
	std::cout << "totl rb parts:" << get_bodies_numparts() << "\n";
	return parts.size() + boundary_parts.size() + get_bodies_numparts();
}


void BuoyancyTest::ODE_near_callback(void *data, dGeomID o1, dGeomID o2)
{
	const int N = 10;
	dContact contact[N];

	int n = dCollide(o1, o2, N, &contact[0].geom, sizeof(dContact));
	for (int i = 0; i < n; i++) {
		contact[i].surface.mode = dContactBounce;
		contact[i].surface.mu   = dInfinity;
		contact[i].surface.bounce     = 0.0; // (0.0~1.0) restitution parameter
		contact[i].surface.bounce_vel = 0.0; // minimum incoming velocity for bounce
		dJointID c = dJointCreateContact(m_ODEWorld, m_ODEJointGroup, &contact[i]);
		dJointAttach (c, dGeomGetBody(contact[i].geom.g1), dGeomGetBody(contact[i].geom.g2));
	}
}


void
BuoyancyTest::copy_to_array(BufferList &buffers)
{
	float4 *pos = buffers.getData<BUFFER_POS>();
	hashKey *hash = buffers.getData<BUFFER_HASH>();
	float4 *vel = buffers.getData<BUFFER_VEL>();
	particleinfo *info = buffers.getData<BUFFER_INFO>();

	std::cout << "Boundary parts: " << boundary_parts.size() << std::endl;
	for (uint i = 0; i < boundary_parts.size(); ++i) {
		float ht = H - boundary_parts[i](2);
		if (ht < 0)
			ht = 0.0;
		float rho = density(ht, 0);
		vel[i] = make_float4(0, 0, 0, rho);
		info[i] = make_particleinfo(PT_BOUNDARY, 0, i);
		calc_localpos_and_hash(boundary_parts[i], info[i], pos[i], hash[i]);
	}
	uint j = boundary_parts.size();
	std::cout << "Boundary part mass: " << pos[j-1].w << std::endl;

	uint object_particle_counter = 0;
	for (uint k = 0; k < m_bodies.size(); k++) {
		PointVect & rbparts = m_bodies[k]->object->GetParts();
		std::cout << "Rigid body " << k << ": " << rbparts.size() << " particles ";
		for (uint i = 0; i < rbparts.size(); i++) {
			uint ij = i + j;
			float ht = H - rbparts[i](2);
			if (ht < 0)
				ht = 0.0;
			float rho = density(ht, 0);
			rho = physparams()->rho0[0];
			vel[ij] = make_float4(0, 0, 0, rho);
			uint ptype = (uint) PT_BOUNDARY;
			switch (m_bodies[k]->type) {
				case MB_ODE:
					ptype |= FG_MOVING_BOUNDARY | FG_COMPUTE_FORCE;
					break;
				case MB_FORCES_MOVING:
					ptype |= FG_COMPUTE_FORCE | FG_MOVING_BOUNDARY;
					break;
				case MB_MOVING:
					ptype |= FG_MOVING_BOUNDARY;
					break;
			}
			info[ij] = make_particleinfo(ptype, k, ij);
			calc_localpos_and_hash(rbparts[i], info[ij], pos[ij], hash[ij]);
		}
		if (k < simparams()->numforcesbodies) {
			gdata->s_hRbFirstIndex[k] = -j + object_particle_counter;
			gdata->s_hRbLastIndex[k] = object_particle_counter + rbparts.size() - 1;
			object_particle_counter += rbparts.size();
		}
		j += rbparts.size();
		std::cout << ", part mass: " << pos[j-1].w << "\n";
	}

	std::cout << "Fluid parts: " << parts.size() << std::endl;
	for (uint i = 0; i < parts.size(); ++i) {
		uint ij = i+j;
		float ht = H - parts[i](2);
		if (ht < 0)
			ht = 0.0;
		float rho = density(ht, 0);
		vel[ij] = make_float4(0, 0, 0, rho);
		info[ij] = make_particleinfo(PT_FLUID, 0, ij);
		calc_localpos_and_hash(parts[i], info[ij], pos[ij], hash[ij]);
	}
	j += parts.size();

	std::cout << "Fluid part mass: " << pos[j-1].w << std::endl;

	std::flush(std::cout);
}
