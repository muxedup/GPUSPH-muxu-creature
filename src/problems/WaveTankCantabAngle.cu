#include "hip/hip_runtime.h"
/*  Copyright 2011 Alexis Herault, Giuseppe Bilotta, Robert A. Dalrymple, Eugenio Rustico, Ciro Del Negro

	Istituto de Nazionale di Geofisica e Vulcanologia
          Sezione di Catania, Catania, Italy

    Universita di Catania, Catania, Italy

    Johns Hopkins University, Baltimore, MD

    This file is part of GPUSPH.

    GPUSPH is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUSPH is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUSPH.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <cmath>
#include <iostream>
#include <stdexcept>

#include "WaveTankCantabAngle.h"
#include "particledefine.h"
#include "GlobalData.h"
#include "cudasimframework.cu"


#define MK_par 2

WaveTankCantabAngle::WaveTankCantabAngle(GlobalData *_gdata) : Problem(_gdata)
{
	// Size and origin of the simulation domain
	lx = 22.;
	ly = 32.0;
	lz = 3.5;
	
	// Data for problem setup
	slope_length = 10.2;  // horizontal extent of tank covered by sloping beach
	h_length = 10;
	height = 2.5;
	beta = .1974;
	
	
    
	SETUP_FRAMEWORK(
	    //viscosity<ARTVISC>,
		//viscosity<KINEMATICVISC>,
		viscosity<SPSVISC>,
		boundary<LJ_BOUNDARY>
		//boundary<MK_BOUNDARY>
	);
	
	m_size = make_double3(lx, ly, lz + 2.0*height);
	m_origin = make_double3(0.0, 0.0, -1.0*height);	 
	 
	addFilter(SHEPARD_FILTER, 20);
	  //MLS_FILTER

	//    Here we have npaddles paddles for a directional seastate
 
	// Add objects to the tank
    use_cyl = false;
	use_cone = false;

	// use a plane for the bottom
	use_bottom_plane = true; 

	// SPH parameters
	set_deltap(0.15);  //0.005f;
	printf("deltap = %f \n", m_deltap);
	m_simparams->dt = 0.0001;
	m_simparams->dtadaptfactor = 0.2;
	m_simparams->buildneibsfreq = 10;
	m_simparams->tend = 55.0; //seconds

//	m_simparams->vorticity = false;
	//Testpoints
//	m_simparams->testpoints = false;

	// Free surface detection
//	m_simparams->surfaceparticle = false;
//	m_simparams->savenormals = false;
/*
	//WaveGage
	add_gage(5., 16.);
	add_gage(14., 16.);
	add_gage(5.,8.); 
	add_gage(14.,8.);

*/
 

    // Physical parameters
	H = 1.0;
	m_physparams->gravity = make_float3(0.0, 0.0, -9.81);
	float g = length(m_physparams->gravity);

	float r0 = m_deltap;
	m_physparams->r0 = r0;
	
	add_fluid(1000.f);
	set_kinematic_visc(0,1.0e-6);
	
    m_physparams->artvisccoeff= 0.2f;
	m_physparams->smagfactor = 0.12*0.12*m_deltap*m_deltap;
	m_physparams->kspsfactor = (2.0/3.0)*0.0066*m_deltap*m_deltap;
	m_physparams->epsartvisc = 0.01*m_simparams->slength*m_simparams->slength;

	// BC when using LJ
	m_physparams->dcoeff = 5.0*g*H;
    //set p1coeff,p2coeff, epsxsph here if different from 12.,6., 0.5

	// BC when using MK
	m_physparams->MK_K = g*H;
	m_physparams->MK_d = 1.1*m_deltap/MK_par;
	m_physparams->MK_beta = MK_par;

	//Wave paddle definition:  location, start & stop times, stroke and frequency (2 \pi/period)
	float waveangle=(3.1415927/180.)*8.0;
	// lateral wavenumber k sin(theta)
	float k=0.58;  //fix for every case
	lambda=k*sin(waveangle);
    npaddles= 16;
	paddle_width= ly/npaddles;	 
	paddle_length = 1.5; //vertical 
	paddle_origin = make_double3(1.0,0.0, 0.);
	paddle_tstart = 0.2;
	paddle_tend = m_simparams->tend;
	// The stroke value is given at free surface level H
	float stroke = .75;
	// m_mbamplitude is the maximal angular value par paddle angle
	// Paddle angle is in [-m_mbamplitude, m_mbamplitude]
	paddle_amplitude = atan(stroke/(2.0*(H - paddle_origin.z)));
	paddle_omega = 2.0*M_PI/2.6;		// period T = 3.6145 s
	
		
		//add wave gages to measure feedback in front of each paddle  
	 //   add_gage(mbpaddledata.amplitude+5*r0,(i+0.5)*paddle_width);
  
	// Drawing and saving times
	add_writer(VTKWRITER,.1);
	
	// Name of problem used for directory creation
	m_name = "WaveTankCantabAngle";
}


WaveTankCantabAngle::~WaveTankCantabAngle(void)
{
	release_memory();
}


void WaveTankCantabAngle::release_memory(void)
{
	parts.clear();
	boundary_parts.clear();
//	test_points.clear();
}

void
WaveTankCantabAngle::moving_bodies_callback(const uint index, Object* object, const double t0, const double t1,
			const float3& force, const float3& torque, const KinematicData& initial_kdata,
			KinematicData& kdata, double3& dx, EulerParameters& dr)
{
    // index gives the number of the paddles
    dx= make_double3(0.0);
    kdata.lvel=make_double3(0.0f, 0.0f, 0.0f);
    lambda_y = lambda * paddle_width *(2*index - 1)/2.;
    if (t1> paddle_tstart & t1 < paddle_tend){
       kdata.avel = make_double3(0.0, paddle_amplitude*paddle_omega*sin(lambda_y+paddle_omega*(t1-paddle_tstart)),0.0);
       EulerParameters dqdt = 0.5*EulerParameters(kdata.avel)*kdata.orientation;
       dr = EulerParameters::Identity() + (t1-t0)*dqdt*kdata.orientation.Inverse();
       dr.Normalize();
	   kdata.orientation = kdata.orientation + (t1 - t0)*dqdt;
	   kdata.orientation.Normalize();
	   }
	else {
	   kdata.avel = make_double3(0.0,0.0,0.0);
	   kdata.orientation = kdata.orientation;
	   dr.Identity();
	}
	
}


int WaveTankCantabAngle::fill_parts()
{
    std::cout << "filling particles" <<"\n";
	const float r0 = m_physparams->r0;
	const float br = (m_simparams->boundarytype == MK_BOUNDARY ? m_deltap/MK_par : r0);
	
	boundary_parts.reserve(100);
	paddle_parts.reserve(500);
	parts.reserve(34000);
   std::cout <<"npaddles = " << npaddles <<"\n";
    experiment_box = Cube(Point(0, 0, 0), h_length + slope_length,ly, height);
//	std::cout << "paddle origins: \n" <<"\n";
    // define the paddles
      const float amplitude = -paddle_amplitude ;
    for (uint i=0; i<npaddles; i++) {	
        double y_paddle = paddle_width*i;
        lambda_y = lambda * (y_paddle+ paddle_width/2.); 
		paddle[i] = Rect(Point(paddle_origin.x, paddle_origin.y+y_paddle, paddle_origin.z), Vector(0, paddle_width-r0, 0),
				Vector(paddle_length*sin(amplitude*cos(lambda_y)), 0, paddle_length*cos(amplitude*cos(lambda_y))));
		paddle[i].SetPartMass(m_deltap, m_physparams->rho0[0]);
		paddle[i].Fill(paddle[i].GetParts(), br, true);
		add_moving_body(&paddle[i], MB_MOVING);
	    set_body_cg(&paddle[i], make_double3(paddle_origin.x, paddle_origin.y+y_paddle, paddle_origin.z));
		std::cout << "paddle[" << i <<"] defined \n";
		std::cout << "  at y = " << y_paddle << "\n";
		  
		}
	
    std::cout << "paddles defined" << "\n";

	bottom_rect = Rect(Point(h_length, 0, 0), Vector(0, ly, 0),
			Vector(slope_length/cos(beta), 0.0, slope_length*tan(beta)));
	if (!use_bottom_plane) {
	   bottom_rect.SetPartMass(m_deltap, m_physparams->rho0[0]);
	   bottom_rect.Fill(boundary_parts,br,true);
	   std::cout << "bottom rectangle defined" <<"\n";
	   }
 
	Rect fluid;
	float z = 0;
	int n = 0;
//	const float amplitude = mbpaddledata.amplitude;
//	printf("amplitude = %f\n",mbpaddledata.amplitude);
	printf("r0 = %f\n", r0);
	printf("h_length, slope_length: %f, %f\n", h_length, slope_length);
    
	while (z < H) {
	 	z = n*m_deltap + 1.5*r0;    //z = n*m_deltap + 1.5*r0;
	 	std::cout << "z = " <<z <<"\n";
		 
		for (uint i=0; i< npaddles-1; i++) {
			double arg = lambda*paddle_width*(2*i+1)/2;
		    double amplitude = paddle_amplitude*sin(arg);
		   
            float x = paddle_origin.x + (z - paddle_origin.z)*tan(amplitude) + 1.0*r0/cos(amplitude);
 		 
			float l = h_length + z/tan(beta) - 1.5*r0/sin(beta) - x;
			float y = r0 + paddle_origin.y + paddle_width*i; // paddle corner
		//	printf("i, amplitude: %d, %f \n", i, amplitude);
		//	printf("x, y, z, l: %f, %f, %f, %f \n", x, y, z, l);
			if (l <0) l = 0;
			if (l > h_length+ slope_length -x) l = h_length+slope_length -r0 -x;
			fluid = Rect(Point(x, y, z),
				Vector(0, paddle_width-r0, 0), Vector(l, 0, 0));
			fluid.SetPartMass(m_deltap, m_physparams->rho0[0]);
			fluid.Fill(parts, m_deltap, true);			
		   }
		// last paddle is r0 shorter

			double amplitude= paddle_amplitude;
			float x = paddle_origin.x + (z - paddle_origin.z)*tan(amplitude) + 1.0*r0/cos(amplitude);
			float l = h_length + z/tan(beta) - 1.5*r0/sin(beta) - x;
			float y = r0 + paddle_origin.y + paddle_width*(npaddles-1);
	//		printf("i, amplitude: %d, %f \n", npaddles, amplitude);
		//	printf("x, y, z, l: %f, %f, %f, %f \n", x, y, z, l);
			if (l <0) l = 0;
			if (l > h_length+ slope_length -x) l = h_length+slope_length -r0 -x;
		    fluid = Rect(Point(x,y,z), Vector(0, paddle_width-2*r0,0), Vector(l,0,0));
		    fluid.SetPartMass(m_deltap, m_physparams->rho0[0]);
		    fluid.Fill(parts, m_deltap,true);
		 n++;
	}
	 
	 /*
	if (m_simparams.testpoints) {
		Point pos = Point(0.5748, 0.1799, 0.2564, 0.0);
		test_points.push_back(pos);
		pos = Point(0.5748, 0.2799, 0.2564, 0.0);
		test_points.push_back(pos);
		pos = Point(1.5748, 0.2799, 0.2564, 0.0);
		test_points.push_back(pos);
	}
	*/
	if (use_cyl) {
		Point p[10];
		p[0] = Point(h_length + slope_length/(cos(beta)*10), ly/2., 0);
		p[1] = Point(h_length + slope_length/(cos(beta)*10), ly/6.,  0);
		p[2] = Point(h_length + slope_length/(cos(beta)*10), 5*ly/6, 0);
		p[3] = Point(h_length + slope_length/(cos(beta)*5), 0, 0);
		p[4] = Point(h_length + slope_length/(cos(beta)*5), ly/3, 0);
		p[5] = Point(h_length + slope_length/(cos(beta)*5), 2*ly/3, 0);
		p[6] = Point(h_length + slope_length/(cos(beta)*5), ly, 0);
		p[7] = Point(h_length + 3*slope_length/(cos(beta)*10), ly/6, 0);
		p[8] = Point(h_length + 3*slope_length/(cos(beta)*10), ly/2, 0);
		p[9] = Point(h_length+ 3*slope_length/(cos(beta)*10), 5*ly/6, 0);
		p[10] = Point(h_length+ 4*slope_length/(cos(beta)*10), ly/2, 0);

		for (int i = 0; i < 11; i++) {
			cyl[i] = Cylinder(p[i], Vector(.025, 0, 0), Vector(0, 0, height));
			cyl[i].SetPartMass(m_deltap, m_physparams->rho0[0]);
			cyl[i].FillBorder(boundary_parts, br, false, false);
			cyl[i].Unfill(parts, br);
		}
	}
	if (use_cone) {
		Point p1 = Point(h_length + slope_length/(cos(beta)*10), ly/2, 0);
		cone = Cone(p1,Vector(ly/4, 0.0, 0.0), Vector(ly/10, 0., 0.), Vector(0, 0, height));
		cone.SetPartMass(m_deltap, m_physparams->rho0[0]);
		cone.FillBorder(boundary_parts, br, false, true);
		cone.Unfill(parts, br);
    }
	
	return parts.size() + boundary_parts.size() + get_bodies_numparts() + test_points.size();
}


uint WaveTankCantabAngle::fill_planes()
{
    if (!use_bottom_plane) {
		return 5;
		}
	else {
		return 6;
		} //corresponds to number of planes
}


void WaveTankCantabAngle::copy_planes(double4 *planes)
{
	const float w = m_size.y;
	const float l = h_length + slope_length;

	//  plane is defined as a x + by +c z + d= 0
	planes[0] = make_double4(0, 0, 1.0, 0);   //bottom, where the first three numbers are the normal, and the last is d.
	planes[1] = make_double4(0, 1.0, 0, 0);   //wall
	planes[2] = make_double4(0, -1.0, 0, w); //far wall
 	planes[3] = make_double4(1.0, 0, 0, 0);  //end
 	planes[4] = make_double4(-1.0, 0, 0, l);  //one end
 	if (use_bottom_plane)  {
		planes[5] = make_double4(-sin(beta),0,cos(beta), h_length*sin(beta));  //sloping bottom starting at x=h_length
	}
}


 


void WaveTankCantabAngle::copy_to_array(BufferList &buffers)
{
    float4 *pos = buffers.getData<BUFFER_POS>();
	hashKey *hash = buffers.getData<BUFFER_HASH>();
	float4 *vel = buffers.getData<BUFFER_VEL>();
	particleinfo *info = buffers.getData<BUFFER_INFO>();
	
    int j = 0;
    /*
	if (test_points.size()) {
		//Testpoints
		std::cout << "\nTest points: " << test_points.size() << "\n";
		std::cout << "      " << 0  << "--" << test_points.size() << "\n";
		for (uint i = 0; i < test_points.size(); i++) {
			calc_localpos_and_hash(test_points[i], info[i], pos[i], hash[i]);
			vel[i] = make_float4(0, 0, 0, m_physparams->rho0[0]);
			info[i]= make_particleinfo(TESTPOINTSPART, 0, i);  // first is type, object, 3rd id
		}
		std::cout << "Test point mass:" << pos[j-1].w << "\n";
		j += test_points.size();
	}
*/
	std::cout << "\nBoundary parts: " << boundary_parts.size() << "\n";
		std::cout << "      "<< j  <<"--"<< j+ boundary_parts.size() << "\n";    //FIXED: 0 --> j 
	for (uint i = j; i < j + boundary_parts.size(); i++) {
		calc_localpos_and_hash(boundary_parts[i - j], info[i], pos[i], hash[i]);
		vel[i] = make_float4(0, 0, 0, m_physparams->rho0[0]);
		info[i]= make_particleinfo(PT_BOUNDARY, 0, i);  // first is type, object, 3rd id
	}
    j += boundary_parts.size();
	std::cout << "Boundary part mass:" << pos[j-1].w << "\n";

	std::cout << "\nFluid parts: " << parts.size() << "\n";
	std::cout << "      "<< j  <<"--"<< j + parts.size() << "\n";
	for (uint i = j; i < j + parts.size(); i++) {
		calc_localpos_and_hash(parts[i - j], info[i], pos[i], hash[i]);
		float rho = m_physparams->rho0[0]; // density(H - pos[i].z, 0);
		vel[i] = make_float4(0, 0, 0, rho);
	    info[i]= make_particleinfo(PT_FLUID, 0, i);
	}
	j += parts.size();
	std::cout << "Fluid part mass:" << pos[j-1].w << "\n";
	
    for (uint k = 0; k < m_bodies.size(); k++) {
			PointVect & rbparts = m_bodies[k]->object->GetParts();
			std::cout << "Rigid body " << k << ": " << rbparts.size() << " particles ";
			for (uint i = 0; i < rbparts.size(); i++) {
				uint ij = i + j;
				float ht = H - rbparts[i](2);
				if (ht < 0)
					ht = 0.0;
				float rho = density(ht, 0);
				rho = m_physparams->rho0[0];
				vel[ij] = make_float4(0, 0, 0, rho);
				uint ptype = (uint) PT_BOUNDARY;
				switch (m_bodies[k]->type) {
					case MB_ODE:
						ptype |= FG_COMPUTE_FORCE | FG_MOVING_BOUNDARY;
						break;
					case MB_FORCES_MOVING:
						ptype |= FG_COMPUTE_FORCE | FG_MOVING_BOUNDARY;
						break;
					case MB_MOVING:
						ptype |= FG_MOVING_BOUNDARY;
						break;
				}
				info[ij] = make_particleinfo(ptype, k, i );
				calc_localpos_and_hash(rbparts[i], info[ij], pos[ij], hash[ij]);
			}
			j += rbparts.size();
			std::cout << ", part mass: " << pos[j-1].w << "\n";
			std::cout << ", part type: " << type(info[j-1])<< "\n";
		}




	std::cout << "Everything uploaded" <<"\n";
}
 

#undef MK_par
