#include "hip/hip_runtime.h"
/*  Copyright 2011-2013 Alexis Herault, Giuseppe Bilotta, Robert A. Dalrymple, Eugenio Rustico, Ciro Del Negro

    Istituto Nazionale di Geofisica e Vulcanologia
        Sezione di Catania, Catania, Italy

    Università di Catania, Catania, Italy

    Johns Hopkins University, Baltimore, MD

    This file is part of GPUSPH.

    GPUSPH is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUSPH is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUSPH.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <math.h>
#include <iostream>

#include "StillWater.h"
#include "GlobalData.h"
#include "cudasimframework.cu"

#define CENTER_DOMAIN 1
// set to coords (x,y,z) if more accuracy is needed in such point
// (waiting for relative coordinates)
#if CENTER_DOMAIN
#define OFFSET_X (-l/2)
#define OFFSET_Y (-w/2)
#define OFFSET_Z (-h/2)
#else
#define OFFSET_X 0
#define OFFSET_Y 0
#define OFFSET_Z 0
#endif

StillWater::StillWater(GlobalData *_gdata) : Problem(_gdata)
{
	m_usePlanes = get_option("use-planes", false); // --use-planes true to enable use of planes for boundaries
	const int mlsIters = get_option("mls", 0); // --mls N to enable MLS filter every N iterations
	const int ppH = get_option("ppH", 16); // --ppH N to change deltap to H/N

	// density diffusion terms: 0 none, 1 Molteni & Colagrossi, 2 Ferrari
	const int rhodiff = get_option("density-diffusion", 1);

	SETUP_FRAMEWORK(
		//viscosity<KINEMATICVISC>,
		viscosity<DYNAMICVISC>,
		//viscosity<ARTVISC>,
		boundary<DYN_BOUNDARY>
		//boundary<SA_BOUNDARY>
		//boundary<LJ_BOUNDARY>
	).select_options(
		rhodiff, FlagSwitch<ENABLE_NONE, ENABLE_DENSITY_DIFFUSION, ENABLE_FERRARI>(),
		m_usePlanes, add_flags<ENABLE_PLANES>()
	);

	if (mlsIters > 0)
		addFilter(MLS_FILTER, mlsIters);

	H = 1;

	set_deltap(H/ppH);

	l = w = sqrt(2)*H; h = 1.1*H;

	// Size and origin of the simulation domain
	m_size = make_double3(l, w ,h);
	m_origin = make_double3(OFFSET_X, OFFSET_Y, OFFSET_Z);

	// SPH parameters
	simparams()->dt = 0.00004f;
	simparams()->dtadaptfactor = 0.3;
	simparams()->buildneibsfreq = 20;
	simparams()->ferrariLengthScale = H;

	// enlarge the domain to take into account the extra layers of particles
	// of the boundary
	if (simparams()->boundarytype == DYN_BOUNDARY && !m_usePlanes) {
		// number of layers
		dyn_layers = ceil(simparams()->kernelradius*simparams()->sfactor);
		// extra layers are one less (since other boundary types still have
		// one layer)
		double3 extra_offset = make_double3((dyn_layers-1)*m_deltap);
		m_origin -= extra_offset;
		m_size += 2*extra_offset;
	} else {
		dyn_layers = 1;
	}

	simparams()->tend = 100.0;
	if (simparams()->boundarytype == SA_BOUNDARY) {
		simparams()->maxneibsnum = 256; // needed during gamma initialization phase
	};

	// Physical parameters
	physparams()->gravity = make_float3(0.0, 0.0, -9.81f);
	const float g = length(physparams()->gravity);
	const float maxvel = sqrt(2*g*H);
	// purely for cosmetic reason, let's round the soundspeed to the next
	// integer
	const float c0 = ceil(10*maxvel);
	add_fluid(1000.0);
	set_equation_of_state(0,  7.0f, c0);

	physparams()->dcoeff = 5.0f*g*H;

	physparams()->r0 = m_deltap;
	//physparams()->visccoeff = 0.05f;
	set_kinematic_visc(0, 3.0e-2f);
	//set_kinematic_visc(0, 1.0e-6f);
	physparams()->artvisccoeff = 0.3f;
	physparams()->epsartvisc = 0.01*simparams()->slength*simparams()->slength;
	physparams()->epsxsph = 0.5f;

	// Drawing and saving times
	add_writer(VTKWRITER, 1.0);

	// Name of problem used for directory creation
	m_name = "StillWater";
}


StillWater::~StillWater(void)
{
	release_memory();
}


void StillWater::release_memory(void)
{
	parts.clear();
	boundary_parts.clear();
}


int StillWater::fill_parts()
{
	// distance between fluid box and wall
	float wd = physparams()->r0;

	parts.reserve(14000);

	experiment_box = Cube(Point(m_origin), m_size.x, m_size.y, m_size.z);

	experiment_box.SetPartMass(wd, physparams()->rho0[0]);

	if (!m_usePlanes) {
		switch (simparams()->boundarytype) {
		case SA_BOUNDARY:
			experiment_box.FillBorder(boundary_parts, boundary_elems, vertex_parts, vertex_indexes, wd, false);
			break;
		case DYN_BOUNDARY:
			experiment_box.FillIn(boundary_parts, m_deltap, dyn_layers, false);
			break;
		default:
			experiment_box.FillBorder(boundary_parts, wd, false);
			break;
		}
	}

	m_fluidOrigin = m_origin;
	if (dyn_layers > 1) // shift by the extra offset of the experiment box
		m_fluidOrigin += make_double3((dyn_layers-1)*m_deltap);
	m_fluidOrigin += make_double3(wd); // one wd space from the boundary
	Cube fluid = Cube(m_fluidOrigin, l-2*wd, w-2*wd, H-2*wd);
	fluid.SetPartMass(m_deltap, physparams()->rho0[0]);
	fluid.Fill(parts, m_deltap);

	//DEBUG: set only one fluid particle
//	parts.clear();
//	parts.push_back(Point(0.0, w/2.f, 0.0));
//	for(int i=0; i < vertex_parts.size(); i++)
//		if(	vertex_parts[i](2) == 0 &&
//			vertex_parts[i](0) > 0.5*w && vertex_parts[i](0) < 0.5*w+2*m_deltap &&
//			vertex_parts[i](1) > 0.5*w && vertex_parts[i](1) < 0.5*w+2*m_deltap)
//			parts.push_back(Point(vertex_parts[i](0) + 0.5*m_deltap, vertex_parts[i](1) + 0.5*m_deltap, 0.0));

	return parts.size() + boundary_parts.size() + vertex_parts.size();
}

void StillWater::copy_planes(PlaneList& planes)
{
	if (!m_usePlanes) return;

	planes.push_back( implicit_plane(0, 0, 1.0, -m_origin.z) );
	planes.push_back( implicit_plane(0, 1.0, 0, -m_origin.x) );
	planes.push_back( implicit_plane(0, -1.0, 0, m_origin.x + w) );
	planes.push_back( implicit_plane(1.0, 0, 0, -m_origin.y) );
	planes.push_back( implicit_plane(-1.0, 0, 0, m_origin.y + l) );
}


void StillWater::copy_to_array(BufferList &buffers)
{
	float4 *pos = buffers.getData<BUFFER_POS>();
	hashKey *hash = buffers.getData<BUFFER_HASH>();
	float4 *vel = buffers.getData<BUFFER_VEL>();
	particleinfo *info = buffers.getData<BUFFER_INFO>();
	vertexinfo *vertices = buffers.getData<BUFFER_VERTICES>();
	float4 *boundelm = buffers.getData<BUFFER_BOUNDELEMENTS>();

	std::cout << "Boundary parts: " << boundary_parts.size() << "\n";
	for (uint i = 0; i < boundary_parts.size(); i++) {
#if 1
		double water_column = m_fluidOrigin.z + H - boundary_parts[i](2);
		if (water_column < 0)
			water_column = 0;
		float rho = density(water_column, 0);
#else
		float rho = physparams()->rho0[0];
#endif
		vel[i] = make_float4(0, 0, 0, rho);
		info[i] = make_particleinfo(PT_BOUNDARY, 0, i);
		calc_localpos_and_hash(boundary_parts[i], info[i], pos[i], hash[i]);
	}
	int j = boundary_parts.size();
	std::cout << "Boundary part mass: " << pos[j-1].w << "\n";

	std::cout << "Fluid parts: " << parts.size() << "\n";
	for (uint i = j; i < j + parts.size(); i++) {
		double water_column = m_fluidOrigin.z + H - parts[i - j](2);
		if (water_column < 0)
			water_column = 0;
		float rho = density(water_column, 0);
		vel[i] = make_float4(0, 0, 0, rho);
		info[i] = make_particleinfo(PT_FLUID, 0, i);
		calc_localpos_and_hash(parts[i-j], info[i], pos[i], hash[i]);
	}
	j += parts.size();
	std::cout << "Fluid part mass: " << pos[j-1].w << "\n";

	if (simparams()->boundarytype == SA_BOUNDARY) {
			uint j = parts.size() + boundary_parts.size();

			std::cout << "Vertex parts: " << vertex_parts.size() << "\n";
		for (uint i = j; i < j + vertex_parts.size(); i++) {
			float rho = density(H - vertex_parts[i-j](2), 0);
			vel[i] = make_float4(0, 0, 0, rho);
			info[i] = make_particleinfo(PT_VERTEX, 0, i);
			calc_localpos_and_hash(vertex_parts[i-j], info[i], pos[i], hash[i]);
		}
		j += vertex_parts.size();
		std::cout << "Vertex part mass: " << pos[j-1].w << "\n";

		if(vertex_indexes.size() != boundary_parts.size()) {
			std::cout << "ERROR! Incorrect connectivity array!\n";
			exit(1);
		}
		if(boundary_elems.size() != boundary_parts.size()) {
			std::cout << "ERROR! Incorrect boundary elements array!\n";
			exit(1);
		}

		uint offset = parts.size() + boundary_parts.size();
		for (uint i = 0; i < boundary_parts.size(); i++) {
			vertex_indexes[i].x += offset;
			vertex_indexes[i].y += offset;
			vertex_indexes[i].z += offset;

			vertices[i] = vertex_indexes[i];

			boundelm[i] = make_float4(boundary_elems[i]);
		}
	}
}
