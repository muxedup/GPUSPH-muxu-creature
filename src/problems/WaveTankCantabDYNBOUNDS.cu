#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include <stdexcept>

#include "WaveTankCantabDYNBOUNDS.h"
#include "particledefine.h"
#include "GlobalData.h"
#include "cudasimframework.cu"

#define OFFSET_X (-lx/2)
#define OFFSET_Y (-ly/2)
#define OFFSET_Z (-lz/2)

using namespace std;

WaveTankCantabDYNBOUNDS::WaveTankCantabDYNBOUNDS(GlobalData *_gdata) : Problem(_gdata)
{
	SETUP_FRAMEWORK(
		viscosity<SPSVISC>,
		boundary<DYN_BOUNDARY>,
		flags<ENABLE_DTADAPT>
	);
	
	// Size and origin of the simulation domain
	lx = 22.;
	ly = 32.0;
	lz = 3.5;
	
	// Data for problem setup
	slope_length = 10.2;  // horizontal extent of tank covered by sloping beach
	h_length = 10;
	height = 2.5;
	beta = .1974;
	
	H = 1;
	
	m_size = make_double3(lx, ly, lz);
	m_origin = make_double3(OFFSET_X, OFFSET_Y, OFFSET_Z);
	
	m_usePlanes = false;
	use_bottom_plane = false;
	
	set_deltap(0.15);
	cout << "deltap = " << m_deltap << endl;
	
	if (m_simparams->boundarytype == DYN_BOUNDARY && !m_usePlanes) {
		// number of layers
		dyn_layers = ceil(m_simparams->kernelradius*m_simparams->sfactor);
		// extra layers are one less (since other boundary types still have
		// one layer)
		double3 extra_offset = make_double3((dyn_layers-1)*m_deltap);
		m_origin -= extra_offset;
		m_size += 2*extra_offset;
	}
	
	// Wave maker parameters
	int numPeriods = 36;
	float paddlePeriod = 2.4;
	float incidentAngle = 0;
	m_simparams->tend = numPeriods * paddlePeriod;
	
	// SPH Stuff
	m_simparams->dt = 2e-5;
	m_simparams->dtadaptfactor = 0.2;
	m_simparams->buildneibsfreq = 10;
	
	// Physical parameters
	m_physparams->gravity = make_float3(0, 0, -9.81);
	const float g = length(m_physparams->gravity);
	const float r0 = m_deltap;
	m_physparams->r0 = r0;
	
	add_fluid(1000.f);
	set_kinematic_visc(0,1.0e-6);
	
    m_physparams->artvisccoeff= 0.2f;
	m_physparams->smagfactor = 0.12*0.12*m_deltap*m_deltap;
	m_physparams->kspsfactor = (2.0/3.0)*0.0066*m_deltap*m_deltap;
	m_physparams->epsartvisc = 0.01*m_simparams->slength*m_simparams->slength;
	
	add_writer(VTKWRITER, 0.1);
	m_name = "WaveTankCantabDYNBOUNDS";
	
}

WaveTankCantabDYNBOUNDS::~WaveTankCantabDYNBOUNDS(void) {
	release_memory();
}

void WaveTankCantabDYNBOUNDS::release_memory(void) {
	parts.clear();
	boundary_parts.clear();
}

int WaveTankCantabDYNBOUNDS::fill_parts() {
	cout << "filling particles" << endl;
	const float r0 = m_physparams->r0;
	
	boundary_parts.reserve(100);
	experiment_box = Cube(m_origin, m_size.x, m_size.y, m_size.z);
	
	experiment_box.FillIn(boundary_parts, r0, dyn_layers, false);
	return parts.size() + boundary_parts.size();
}

uint WaveTankCantabDYNBOUNDS::fill_planes()
{
	return (m_usePlanes ? 5 : 0);
}

void WaveTankCantabDYNBOUNDS::copy_planes(double4* planes)
{
	if (!m_usePlanes) return;

	planes[0] = make_double4(0, 0, 1.0, -m_origin.z);
	planes[1] = make_double4(0, 1.0, 0, -m_origin.x);
	planes[2] = make_double4(0, -1.0, 0, m_origin.x + ly);
	planes[3] = make_double4(1.0, 0, 0, -m_origin.y);
	planes[4] = make_double4(-1.0, 0, 0, m_origin.y + lz);
}

void WaveTankCantabDYNBOUNDS::copy_to_array(BufferList &buffers) {
	float4 *pos = buffers.getData<BUFFER_POS>();
	hashKey *hash = buffers.getData<BUFFER_HASH>();
	float4 *vel = buffers.getData<BUFFER_VEL>();
	particleinfo *info = buffers.getData<BUFFER_INFO>();
	vertexinfo *vertices = buffers.getData<BUFFER_VERTICES>();
	float4 *boundelm = buffers.getData<BUFFER_BOUNDELEMENTS>();

	cout << "Boundary parts: " << boundary_parts.size() << "\n";
	for (uint i = 0; i < boundary_parts.size(); i++) {
#if 1
		double water_column = H - boundary_parts[i](2);
		if (water_column < 0)
			water_column = 0;
		float rho = density(water_column, 0);
#else
		float rho = m_physparams->rho0[0];
#endif
		vel[i] = make_float4(0, 0, 0, rho);
		info[i] = make_particleinfo(PT_BOUNDARY, 0, i);
		calc_localpos_and_hash(boundary_parts[i], info[i], pos[i], hash[i]);
	}
	int j = boundary_parts.size();
	cout << "Boundary part mass: " << pos[j-1].w << "\n";
	cout << "Everything Uploaded" << endl;
}