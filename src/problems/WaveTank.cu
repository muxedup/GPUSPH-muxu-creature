#include "hip/hip_runtime.h"
/*  Copyright 2011-2013 Alexis Herault, Giuseppe Bilotta, Robert A. Dalrymple, Eugenio Rustico, Ciro Del Negro

    Istituto Nazionale di Geofisica e Vulcanologia
        Sezione di Catania, Catania, Italy

    Università di Catania, Catania, Italy

    Johns Hopkins University, Baltimore, MD

    This file is part of GPUSPH.

    GPUSPH is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUSPH is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUSPH.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <cmath>
#include <iostream>
#include <stdexcept>

#include "WaveTank.h"
#include "particledefine.h"
#include "GlobalData.h"
#include "cudasimframework.cu"


#define MK_par 2

WaveTank::WaveTank(GlobalData *_gdata) : Problem(_gdata)
{
	// Size and origin of the simulation domain
	lx = 9.0;
	ly = 0.6;
	lz = 1.0;


	// Data for problem setup
	slope_length = 8.5;
	h_length = 0.5;
	height = .63;
	beta = 4.2364*M_PI/180.0;

	SETUP_FRAMEWORK(
	    //viscosity<ARTVISC>,
		//viscosity<KINEMATICVISC>,
		viscosity<SPSVISC>,
		boundary<LJ_BOUNDARY>
		//boundary<MK_BOUNDARY>
	);

	m_size = make_double3(lx, ly, lz + 2.0*height);
	m_origin = make_double3(0.0, 0.0, -2.0*height);

	addFilter(SHEPARD_FILTER, 20);
	  //MLS_FILTER


	// Add objects to the tank
	use_cyl = false;
	use_cone = false;

	// use a plane for the bottom
	use_bottom_plane = 1;  //1 for plane; 0 for particles

	// SPH parameters
	set_deltap(0.03f);  //0.005f;
	m_simparams->dt = 0.0001;
	m_simparams->dtadaptfactor = 0.2;
	m_simparams->buildneibsfreq = 10;
	m_simparams->tend = 10.0f; //seconds

	//WaveGage
//	add_gage(1, 0.3);
//	add_gage(0.5, 0.3);

	// Physical parameters
	H = 0.45;
	m_physparams->gravity = make_float3(0.0f, 0.0f, -9.81f);
	float g = length(m_physparams->gravity);

	float r0 = m_deltap;
	m_physparams->r0 = r0;

	add_fluid( 1000.0f);
	set_equation_of_state(0,  7.0f, 20.f);
	set_kinematic_visc(0,1.0e-6);

	m_physparams->artvisccoeff =  0.2;
	m_physparams->smagfactor = 0.12*0.12*m_deltap*m_deltap;
	m_physparams->kspsfactor = (2.0/3.0)*0.0066*m_deltap*m_deltap;
	m_physparams->epsartvisc = 0.01*m_simparams->slength*m_simparams->slength;

	// BC when using LJ
	m_physparams->dcoeff = 0.5*g*H;
	//set p1coeff,p2coeff, epsxsph here if different from 12.,6., 0.5

	// BC when using MK
	m_physparams->MK_K = g*H;
	m_physparams->MK_d = 1.1*m_deltap/MK_par;
	m_physparams->MK_beta = MK_par;

	//Wave paddle definition:  location, start & stop times, stroke and frequency (2 \pi/period)

	paddle_length = .7f;
	paddle_width = m_size.y - 2*r0;
	paddle_tstart=0.5f;
	paddle_origin = make_double3(0.25f, r0, 0.0f);
	paddle_tend = 30.0f;//seconds
	// The stroke value is given at free surface level H
	float stroke = 0.2;
	// m_mbamplitude is the maximal angular value for paddle angle
	// Paddle angle is in [-m_mbamplitude, m_mbamplitude]
	paddle_amplitude = atan(stroke/(2.0*(H - paddle_origin.z)));
	std::cout << "\npaddle_amplitude (radians): " << paddle_amplitude << "\n";
	paddle_omega = 2.0*M_PI/0.8;		// period T = 0.8 s

	// Drawing and saving times

	add_writer(VTKWRITER, .1);  //second argument is saving time in seconds

	// Name of problem used for directory creation
	m_name = "WaveTank";
}


WaveTank::~WaveTank(void)
{
	release_memory();
}


void WaveTank::release_memory(void)
{
	parts.clear();
	boundary_parts.clear();
}

void
WaveTank::moving_bodies_callback(const uint index, Object* object, const double t0, const double t1,
			const float3& force, const float3& torque, const KinematicData& initial_kdata,
			KinematicData& kdata, double3& dx, EulerParameters& dr)
{

    dx= make_double3(0.0);
    kdata.lvel=make_double3(0.0f, 0.0f, 0.0f);
    if (t1> paddle_tstart & t1 < paddle_tend){
       kdata.avel = make_double3(0.0, paddle_amplitude*paddle_omega*sin(paddle_omega*(t1-paddle_tstart)),0.0);
       EulerParameters dqdt = 0.5*EulerParameters(kdata.avel)*kdata.orientation;
       dr = EulerParameters::Identity() + (t1-t0)*dqdt*kdata.orientation.Inverse();
       dr.Normalize();
	   kdata.orientation = kdata.orientation + (t1 - t0)*dqdt;
	   kdata.orientation.Normalize();
	   }
	else {
	   kdata.avel = make_double3(0.0,0.0,0.0);
	   kdata.orientation = kdata.orientation;
	   dr.Identity();
	}
}


int WaveTank::fill_parts()
{
	const float r0 = m_physparams->r0;
	const float br = (m_simparams->boundarytype == MK_BOUNDARY ? m_deltap/MK_par : r0);

	experiment_box = Cube(Point(0, 0, 0), h_length + slope_length,ly, height);

	boundary_parts.reserve(100);
	parts.reserve(34000);

    const float amplitude = -paddle_amplitude ;
	paddle = Rect(Point(paddle_origin), Vector(0, paddle_width, 0),
				Vector(paddle_length*sin(amplitude), 0, paddle_length*cos(amplitude)));
    paddle.SetPartMass(m_deltap, m_physparams->rho0[0]);
	paddle.Fill(paddle.GetParts(), br, true);
	add_moving_body(&paddle, MB_MOVING);
	set_body_cg(&paddle, paddle_origin);

	bottom_rect = Rect(Point(h_length, 0, 0), Vector(0, ly, 0),
		//	Vector(slope_length/cos(beta), 0.0, slope_length*tan(beta)));
		 Vector(0.0,0.0,paddle_length));
	if (!use_bottom_plane) {
	   bottom_rect.SetPartMass(m_deltap, m_physparams->rho0[0]);
	   bottom_rect.Fill(boundary_parts,br,true);
	   }

	Rect fluid;
	float z = 0;
	int n = 0;
	while (z < H) {
		z = n*m_deltap + 1.5*r0;    //z = n*m_deltap + 1.5*r0;
		float x = paddle_origin.x + (z - paddle_origin.z)*tan(amplitude) + 1.0*r0/cos(amplitude);
		float l = h_length + z/tan(beta) - 1.5*r0/sin(beta) - x;
		fluid = Rect(Point(x,  r0, z),
				Vector(0, ly-2.0*r0, 0), Vector(l, 0, 0));
		fluid.SetPartMass(m_deltap, m_physparams->rho0[0]);
		fluid.Fill(parts, m_deltap, true);
		n++;
	 }

/*
	if (m_simparams->testpoints) {
		Point pos = Point(0.5748, 0.1799, 0.2564, 0.0);
		test_points.push_back(pos);
		pos = Point(0.5748, 0.2799, 0.2564, 0.0);
		test_points.push_back(pos);
		pos = Point(1.5748, 0.2799, 0.2564, 0.0);
		test_points.push_back(pos);
	}
*/
	if (use_cyl) {
		Point p[10];
		p[0] = Point(h_length + slope_length/(cos(beta)*10), ly/2., 0);
		p[1] = Point(h_length + slope_length/(cos(beta)*10), ly/6.,  0);
		p[2] = Point(h_length + slope_length/(cos(beta)*10), 5*ly/6, 0);
		p[3] = Point(h_length + slope_length/(cos(beta)*5), 0, 0);
		p[4] = Point(h_length + slope_length/(cos(beta)*5), ly/3, 0);
		p[5] = Point(h_length + slope_length/(cos(beta)*5), 2*ly/3, 0);
		p[6] = Point(h_length + slope_length/(cos(beta)*5), ly, 0);
		p[7] = Point(h_length + 3*slope_length/(cos(beta)*10), ly/6, 0);
		p[8] = Point(h_length + 3*slope_length/(cos(beta)*10), ly/2, 0);
		p[9] = Point(h_length+ 3*slope_length/(cos(beta)*10), 5*ly/6, 0);
		p[10] = Point(h_length+ 4*slope_length/(cos(beta)*10), ly/2, 0);

		for (int i = 0; i < 11; i++) {
			cyl[i] = Cylinder(p[i], Vector(.025, 0, 0), Vector(0, 0, height));
			cyl[i].SetPartMass(m_deltap, m_physparams->rho0[0]);
			cyl[i].FillBorder(boundary_parts, br, false, false);
			cyl[i].Unfill(parts, br);
		}
	}
	if (use_cone) {
		Point p1 = Point(h_length + slope_length/(cos(beta)*10), ly/2, 0);
		cone = Cone(p1,Vector(ly/4, 0.0, 0.0), Vector(ly/10, 0., 0.), Vector(0, 0, height));
		cone.SetPartMass(m_deltap, m_physparams->rho0[0]);
		cone.FillBorder(boundary_parts, br, false, true);
		cone.Unfill(parts, br);
    }

	return  boundary_parts.size() + get_bodies_numparts() +parts.size(); // + test_points.size();
}


uint WaveTank::fill_planes()
{
    if (!use_bottom_plane) {
		return 5;
		}
	else {
		return 6;
		} //corresponds to number of planes
}


void WaveTank::copy_planes(double4 *planes)
{
	const double w = m_size.y;
	const double l = h_length + slope_length;

	//  plane is defined as a x + by +c z + d= 0
	planes[0] = make_double4(0, 0, 1.0, 0);   //bottom, where the first three numbers are the normal, and the last is d.
	planes[1] = make_double4(0, 1.0, 0, 0);   //wall
	planes[2] = make_double4(0, -1.0, 0, w); //far wall
	planes[3] = make_double4(1.0, 0, 0, 0);  //end
	planes[4] = make_double4(-1.0, 0, 0, l);  //one end
	if (use_bottom_plane)  {
		planes[5] = make_double4(-sin(beta),0,cos(beta), h_length*sin(beta));  //sloping bottom starting at x=h_length
	}
}

void WaveTank::copy_to_array(BufferList &buffers)
{
	float4 *pos = buffers.getData<BUFFER_POS>();
	hashKey *hash = buffers.getData<BUFFER_HASH>();
	float4 *vel = buffers.getData<BUFFER_VEL>();
	particleinfo *info = buffers.getData<BUFFER_INFO>();

	int j = 0;
	/*
	if (test_points.size()) {
		//Testpoints
		std::cout << "\nTest points: " << test_points.size() << "\n";
		std::cout << "      " << j << "--" << test_points.size() << "\n";
		for (uint i = 0; i < test_points.size(); i++) {
			vel[i] = make_float4(0, 0, 0, m_physparams->rho0[0]);
			info[i]= make_particleinfo(TESTPOINTSPART, 0, i);  // first is type, object, 3rd id
			calc_localpos_and_hash(test_points[i], info[i], pos[i], hash[i]);
		}
		j += test_points.size();
		std::cout << "Test point mass:" << pos[j-1].w << "\n";
	}
	*/

	std::cout << "\nBoundary parts: " << boundary_parts.size() << "\n";
	std::cout << "      " << j  << "--" << boundary_parts.size() << "\n";
	for (uint i = j; i < j + boundary_parts.size(); i++) {
		vel[i] = make_float4(0, 0, 0, m_physparams->rho0[0]);
		info[i]= make_particleinfo(PT_BOUNDARY, 0, i);  // first is type, object, 3rd id
		calc_localpos_and_hash(boundary_parts[i-j], info[i], pos[i], hash[i]);
	}
	j += boundary_parts.size();
	std::cout << "Boundary part mass:" << pos[j-1].w << "\n";

	uint object_particle_counter = 0;
	for (uint k = 0; k < m_bodies.size(); k++) {
			PointVect & rbparts = m_bodies[k]->object->GetParts();
			std::cout << "Rigid body " << k << ": " << rbparts.size() << " particles ";
			for (uint i = 0; i < rbparts.size(); i++) {
				uint ij = i + j;
				float ht = H - rbparts[i](2);
				if (ht < 0)
					ht = 0.0;
				float rho = density(ht, 0);
				rho = m_physparams->rho0[0];
				vel[ij] = make_float4(0, 0, 0, rho);
				uint ptype = (uint) PT_BOUNDARY;
				switch (m_bodies[k]->type) {
					case MB_ODE:
						ptype |= FG_COMPUTE_FORCE | FG_MOVING_BOUNDARY;
						break;
					case MB_FORCES_MOVING:
						ptype |= FG_COMPUTE_FORCE | FG_MOVING_BOUNDARY;
						break;
					case MB_MOVING:
						ptype |= FG_MOVING_BOUNDARY;
						break;
				}
				info[ij] = make_particleinfo(ptype, k, ij);
				calc_localpos_and_hash(rbparts[i], info[ij], pos[ij], hash[ij]);
			}
			if (k < m_simparams->numforcesbodies) {
				gdata->s_hRbFirstIndex[k] = -j + object_particle_counter;
				gdata->s_hRbLastIndex[k] = object_particle_counter + rbparts.size() - 1;
				object_particle_counter += rbparts.size();
			}
			j += rbparts.size();
			std::cout << ", part mass: " << pos[j-1].w << "\n";
			std::cout << ", part type: " << type(info[j-1])<< "\n";
	}

	std::cout << "\nFluid parts: " << parts.size() << "\n";
	std::cout << "      "<< j  << "--" << j + parts.size() << "\n";
	for (uint i = j; i < j + parts.size(); i++) {
		vel[i] = make_float4(0, 0, 0, m_physparams->rho0[0]);
		info[i]= make_particleinfo(PT_FLUID, 0, i);
		calc_localpos_and_hash(parts[i-j], info[i], pos[i], hash[i]);
	}
	j += parts.size();
	std::cout << "Fluid part mass:" << pos[j-1].w << "\n";

	std::cout << "Everything uploaded" <<"\n";
}


#undef MK_par
