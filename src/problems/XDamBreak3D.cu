#include "hip/hip_runtime.h"
/*  Copyright 2011-2013 Alexis Herault, Giuseppe Bilotta, Robert A. Dalrymple, Eugenio Rustico, Ciro Del Negro

    Istituto Nazionale di Geofisica e Vulcanologia
        Sezione di Catania, Catania, Italy

    Università di Catania, Catania, Italy

    Johns Hopkins University, Baltimore, MD

    This file is part of GPUSPH.

    GPUSPH is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUSPH is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUSPH.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <cmath>
#include <iostream>

#include "XDamBreak3D.h"
#include "Cube.h"
#include "Point.h"
#include "Vector.h"
#include "GlobalData.h"
#include "cudasimframework.cu"

XDamBreak3D::XDamBreak3D(GlobalData *_gdata) : XProblem(_gdata)
{
	// *** user parameters from command line
	const bool WET = get_option("wet", false);
	const bool USE_PLANES = get_option("use_planes", true);
	const uint NUM_OBSTACLES = get_option("num_obstacles", 1);
	const bool ROTATE_OBSTACLE = get_option("rotate_obstacle", true);
	const uint NUM_TESTPOINTS = get_option("num_testpoints", 3);
	// density diffusion terms: 0 none, 1 Molteni & Colagrossi, 2 Ferrari
	const int RHODIFF = get_option("density-diffusion", 1);

	// ** framework setup
	// viscosities: ARTVISC*, KINEMATICVISC*, DYNAMICVISC*, SPSVISC, KEPSVISC
	// boundary types: LJ_BOUNDARY*, MK_BOUNDARY, SA_BOUNDARY, DYN_BOUNDARY*
	// * = tested in thsi problem
	SETUP_FRAMEWORK(
		viscosity<ARTVISC>,
		boundary<LJ_BOUNDARY>
	).select_options(
		RHODIFF, FlagSwitch<ENABLE_NONE, ENABLE_DENSITY_DIFFUSION, ENABLE_FERRARI>(),
		USE_PLANES, add_flags<ENABLE_PLANES>()
	);

	// Allow user to set the MLS frequency at runtime. Default to 0 if density
	// diffusion is enabled or Ferrari correction is enabled, 10 otherwise
	const int mlsIters = get_option("mls",
		(simparams()->simflags & (ENABLE_DENSITY_DIFFUSION | ENABLE_FERRARI)) ? 0 : 10);

	if (mlsIters > 0)
		addFilter(MLS_FILTER, mlsIters);

	// Explicitly set number of layers. Also, prevent having undefined number of layers before the constructor ends.
	setDynamicBoundariesLayers(3);

	// *** Initialization of minimal physical parameters
	set_deltap(0.02f);
	physparams()->r0 = m_deltap;
	physparams()->gravity = make_float3(0.0, 0.0, -9.81);
	const float g = length(physparams()->gravity);
	const double H = 0.4;
	physparams()->dcoeff = 5.0f * g * H;
	add_fluid(1000.0);
	set_equation_of_state(0,  7.0f, 20.0f);
	//set_kinematic_visc(0, 1.0e-2f);
	set_dynamic_visc(0, 1.0e-4f);

	// *** Initialization of minimal simulation parameters
	simparams()->maxneibsnum = 128 + 64;

	// *** Other parameters and settings
	add_writer(VTKWRITER, 0.005f);
	m_name = "XDamBreak3D";

	// *** Geometrical parameters, starting from the size of the domain
	const double dimX = 1.6;
	const double dimY = 0.67;
	const double dimZ = 0.6;
	const double obstacle_side = 0.12;
	const double obstacle_xpos = 0.9;
	const double water_length = 0.4;
	const double water_height = H;
	const double water_bed_height = 0.1;

	// If we used only makeUniverseBox(), origin and size would be computed automatically
	m_origin = make_double3(0, 0, 0);
	m_size = make_double3(dimX, dimY, dimZ);

	// set positioning policy to PP_CORNER: given point will be the corner of the geometry
	setPositioning(PP_CORNER);

	// main container
	if (USE_PLANES) {
		// limit domain with 6 planes
		makeUniverseBox(m_origin, m_origin + m_size);
	} else {
		GeometryID box =
			addBox(GT_FIXED_BOUNDARY, FT_BORDER, m_origin, dimX, dimY, dimZ);
		// we simulate inside the box, so do not erase anything
		setEraseOperation(box, ET_ERASE_NOTHING);
	}

	// Planes unfill automatically but the box won't, to void deleting all the water. Thus,
	// we define the water at already the right distance from the walls.
	double BOUNDARY_DISTANCE = m_deltap;
	if (simparams()->boundarytype == DYN_BOUNDARY && !USE_PLANES)
			BOUNDARY_DISTANCE *= getDynamicBoundariesLayers();

	// Add the main water part
	addBox(GT_FLUID, FT_SOLID, Point(BOUNDARY_DISTANCE, BOUNDARY_DISTANCE, BOUNDARY_DISTANCE),
		water_length - BOUNDARY_DISTANCE, dimY - 2 * BOUNDARY_DISTANCE, water_height - BOUNDARY_DISTANCE);
	// Add the water bed if wet. After we'll implement the unfill with custom dx, it will be possible to declare
	// the water bed overlapping with the main part.
	if (WET) {
		addBox(GT_FLUID, FT_SOLID,
			Point(water_length + m_deltap, BOUNDARY_DISTANCE, BOUNDARY_DISTANCE),
			dimX - water_length - BOUNDARY_DISTANCE - m_deltap,
			dimY - 2 * BOUNDARY_DISTANCE,
			water_bed_height - BOUNDARY_DISTANCE);
	}

	// set positioning policy to PP_BOTTOM_CENTER: given point will be the center of the base
	setPositioning(PP_BOTTOM_CENTER);

	// add one or more obstacles
	const double Y_DISTANCE = dimY / (NUM_OBSTACLES + 1);
	// rotation angle
	const double Z_ANGLE = M_PI / 4;

	for (uint i = 0; i < NUM_OBSTACLES; i++) {
		// Obstacle is of type GT_MOVING_BODY, although the callback is not even implemented, to
		// make the forces feedback available
		GeometryID obstacle = addBox(GT_MOVING_BODY, FT_BORDER,
			Point(obstacle_xpos, Y_DISTANCE * (i+1) + (ROTATE_OBSTACLE ? obstacle_side/2 : 0), 0),
				obstacle_side, obstacle_side, dimZ );
		if (ROTATE_OBSTACLE) {
			rotate(obstacle, 0, 0, Z_ANGLE);
			// until we'll fix it, the rotation centers are always the corners
			// shift(obstacle, 0, obstacle_side/2, 0);
		}
		// enable force feedback to measure forces
		enableFeedback(obstacle);
	}

	// Optionally, add a floating objects
	/*
	// set positioning policy to PP_CENTER: given point will be the geometrical center of the object
	setPositioning(PP_CENTER);
	GeometryID floating_obj =
		addSphere(GT_FLOATING_BODY, FT_BORDER, Point(water_length, dimY/2, water_height), obstacle_side);
	// half water density to make it float
	setMassByDensity(floating_obj, m_physparams->rho0[0] / 2);
	setParticleMassByDensity(floating_obj, m_physparams->rho0[0] / 2);
	// disable collisions: will only interact with fluid
	// disableCollisions(floating_obj);
	*/

	// add testpoints
	const float TESTPOINT_DISTANCE = dimZ / (NUM_TESTPOINTS + 1);
	for (uint t = 0; t < NUM_TESTPOINTS; t++)
		addTestPoint(Point(dimX, dimY/2.0, t * TESTPOINT_DISTANCE));
}

// since the fluid topology is roughly symmetric along Y through the whole simulation, prefer Y split
void XDamBreak3D::fillDeviceMap()
{
	fillDeviceMapByAxis(Y_AXIS);
}
