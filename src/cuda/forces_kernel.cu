#include "hip/hip_runtime.h"
/*  Copyright 2013 Alexis Herault, Giuseppe Bilotta, Robert A.
 	Dalrymple, Eugenio Rustico, Ciro Del Negro

	Conservatoire National des Arts et Metiers, Paris, France

	Istituto Nazionale di Geofisica e Vulcanologia,
    Sezione di Catania, Catania, Italy

    Universita di Catania, Catania, Italy

    Johns Hopkins University, Baltimore, MD

	This file is part of GPUSPH.

    GPUSPH is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUSPH is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUSPH.  If not, see <http://www.gnu.org/licenses/>.
*/

/*
 * Device code.
 */

#ifndef _FORCES_KERNEL_
#define _FORCES_KERNEL_

#include "particledefine.h"
#include "textures.cuh"
#include "vector_math.h"
#include "multi_gpu_defines.h"
#include "GlobalData.h"

#include "kahan.h"
#include "tensor.cu"

#if __COMPUTE__ < 20
#define printf(...) /* eliminate printf from 1.x */
#endif

// Single-precision M_PI
// FIXME : ah, ah ! Single precision with 976896587958795795 decimals ....
#define M_PIf 3.141592653589793238462643383279502884197169399375105820974944f

#define MAXKASINDEX 10

/** \namespace cuforces
 *  \brief Contains all device functions/kernels/variables used force computations, filters and boundary conditions
 *
 *  The namespace cuforces contains all the device part of force computations, filters and boundary conditions :
 *  	- device constants/variables
 *  	- device functions
 *  	- kernels
 *
 *  \ingroup forces
 */
namespace cuforces {

using namespace hipcubounds;

// Core SPH functions
#include "sph_core_utils.cuh"
#include "gamma.cuh"

/** \name Device constants
 *  @{ */
__constant__ idx_t	d_neiblist_end;			///< maximum number of neighbors * number of allocated particles
__constant__ idx_t	d_neiblist_stride;		///< stride between neighbors of the same particle

__constant__ int	d_numfluids;			///< number of different fluids

__constant__ float	d_sqC0[MAX_FLUID_TYPES];	///< square of sound speed for at-rest density for each fluid

__constant__ float	d_ferrari;				///< coefficient for Ferrari correction
__constant__ float	d_rhodiffcoeff;			///< coefficient for density diffusion

__constant__ float	d_epsinterface;			///< interface epsilon for simplified surface tension in Grenier

// LJ boundary repusion force comuting
__constant__ float	d_dcoeff;
__constant__ float	d_p1coeff;
__constant__ float	d_p2coeff;
__constant__ float	d_r0;

// Monaghan-Kaijar boundary repulsion force constants
__constant__ float	d_MK_K;		///< This is typically the square of the maximum velocity, or gravity times the maximum height
__constant__ float	d_MK_d;		///< This is typically the distance between boundary particles
__constant__ float	d_MK_beta;	///< This is typically the ration between h and the distance between boundary particles

__constant__ float	d_visccoeff[MAX_FLUID_TYPES];	///< viscous coefficient
__constant__ float	d_epsartvisc;					///< epsilon of artificial viscosity

__constant__ float	d_partsurf;		///< particle surface (typically particle spacing suared)

// Sub-Particle Scale (SPS) Turbulence parameters
__constant__ float	d_smagfactor;
__constant__ float	d_kspsfactor;

// Rigid body data
__constant__ int3	d_rbcgGridPos[MAX_BODIES]; //< cell of the center of gravity
__constant__ float3	d_rbcgPos[MAX_BODIES]; //< in-cell coordinate of the center of gravity
__constant__ int	d_rbstartindex[MAX_BODIES];
__constant__ float	d_objectobjectdf;
__constant__ float	d_objectboundarydf;


/*  @} */

/** \name Device functions
 *  @{ */

/************************************************************************************************************/
/*							  Functions used by the different CUDA kernels							        */
/************************************************************************************************************/

//! Lennard-Jones boundary repulsion force
__device__ __forceinline__ float
LJForce(const float r)
{
	float force = 0.0f;

	if (r <= d_r0)
		force = d_dcoeff*(__powf(d_r0/r, d_p1coeff) - __powf(d_r0/r, d_p2coeff))/(r*r);

	return force;
}

//! Monaghan-Kajtar boundary repulsion force
/*!
 Monaghan-Kajtar boundary repulsion force doi:10.1016/j.cpc.2009.05.008
 to be multiplied by r_aj vector
 we allow the fluid particle mass mass_f to be different from the
 boundary particle mass mass_b even though they are typically the same
 (except for multi-phase fluids)
*/
__device__ __forceinline__ float
MKForce(const float r, const float slength,
		const float mass_f, const float mass_b)
{
	// MK always uses the 1D cubic or quintic Wendland spline
	float w = 0.0f;

	float force = 0.0f;

	// Wendland has radius 2
	if (r <= 2*slength) {	//TODO: fixme use influenceradius
		float qq = r/slength;
		w = 1.8f * __powf(1.0f - 0.5f*qq, 4.0f) * (2.0f*qq + 1.0f);  //TODO: optimize
		// float dist = r - d_MK_d;
		float dist = max(d_epsartvisc, r - d_MK_d);
		force = d_MK_K*w*2*mass_b/(d_MK_beta * dist * r * (mass_f+mass_b));
	}

	return force;
}
/************************************************************************************************************/

/***************************************** Viscosities *******************************************************/
//! Artificial viscosity
__device__ __forceinline__ float
artvisc(	const float	vel_dot_pos,
			const float	rho,
			const float	neib_rho,
			const float	sspeed,
			const float	neib_sspeed,
			const float	r,
			const float	slength)
{
	// TODO check if it makes sense to support different artificial viscosity coefficients
	// for different fluids
	return vel_dot_pos*slength*d_visccoeff[0]*(sspeed + neib_sspeed)/
									((r*r + d_epsartvisc)*(rho + neib_rho));
}


// ATTENTION: for all non artificial viscosity
// µ is the dynamic viscosity (ρν)

//! Morris laminar viscous term
/*!
 Scalar part of viscosity using Morris 1997
 expression 21 p218 when all particles have the same viscosity
 in this case d_visccoeff = 4 nu
 returns 4.mj.nu/(ρi + ρj) (1/r ∂Wij/∂r)
*/
__device__ __forceinline__ float
laminarvisc_kinematic(	const float	rho,
						const float	neib_rho,
						const float	neib_mass,
						const float	f)
{
	// NOTE: this won't work in multi-fluid!
	// TODO FIXME kinematic viscosity should probably be marked as incompatible
	// with multi-fluid (or at least if fluids don't have the same, constant
	// viscosity
	return neib_mass*d_visccoeff[0]*f/(rho + neib_rho);
}


//! Morris laminar viscous term for variable viscosity
/*!
 Same behaviour as laminarvisc_kinematic but for particle
 dependent viscosity.
 returns mj.(µi + µi)/(ρi.ρj) (1/r ∂Wij/∂r)
*/
__device__ __forceinline__ float
laminarvisc_dynamic(const float	rho,
					const float	neib_rho,
					const float	neib_mass,
					const float	f,
					const float	visc,
					const float	neib_visc)
{
	return neib_mass*(visc + neib_visc)*f/(rho*neib_rho);
}
/************************************************************************************************************/


/*********************************** Adptative time stepping ************************************************/
// Computes dt across different GPU blocks
/*!
 Function called at the end of the forces or powerlawVisc function doing
 a per block maximum reduction
 cflOffset is used in case the forces kernel was partitioned (striping)
*/
__device__ __forceinline__ void
dtadaptBlockReduce(	float*	sm_max,
					float*	cfl,
					uint	cflOffset)
{
	for(unsigned int s = blockDim.x/2; s > 0; s >>= 1)
	{
		__syncthreads();
		if (threadIdx.x < s)
		{
			sm_max[threadIdx.x] = max(sm_max[threadIdx.x + s], sm_max[threadIdx.x]);
		}
	}

	// write result for this block to global mem
	if (!threadIdx.x)
		cfl[cflOffset + blockIdx.x] = sm_max[0];
}
/************************************************************************************************************/

/******************** Functions for computing repulsive force directly from DEM *****************************/

// TODO: check for the maximum timestep

//! Computes normal and viscous force wrt to solid planar boundary
__device__ __forceinline__ float
PlaneForce(	const int3&		gridPos,
			const float3&	pos,
			const float		mass,
			const plane_t&	plane,
			const float3&	vel,
			const float		dynvisc,
			float4&			force)
{
	// relative position of our particle from the reference point of the plane
	const float r = PlaneDistance(gridPos, pos, plane);
	if (r < d_r0) {
		const float DvDt = LJForce(r);
		// Unitary normal vector of the surface
		const float3 relPos = plane.normal*r;

		as_float3(force) += DvDt*relPos;

		// tangential velocity component
		const float3 v_t = vel - dot(vel, relPos)/r*relPos/r; //TODO: check

		// f = -µ u/∆n

		// viscosity
		// float coeff = -dynvisc*M_PI*(d_r0*d_r0-r*r)/(pos.w*r);
		// float coeff = -dynvisc*M_PI*(d_r0*d_r0*3/(M_PI*2)-r*r)/(pos.w*r);
		const float coeff = -dynvisc*d_partsurf/(mass*r);

		// coeff should not be higher than needed to nil v_t in the maximum allowed dt
		// coefficients are negative, so the smallest in absolute value is the biggest

		/*
		float fmag = length(as_float3(force));
		float coeff2 = -sqrt(fmag/slength)/(d_dtadaptfactor*d_dtadaptfactor);
		if (coeff2 < -d_epsartvisc)
			coeff = max(coeff, coeff2);
			*/

		as_float3(force) += coeff*v_t;

		return -coeff;
	}

	return 0.0f;
}

//! DOC-TODO Describe function
__device__ __forceinline__ float
GeometryForce(	const int3&		gridPos,
				const float3&	pos,
				const float		mass,
				const float3&	vel,
				const float		dynvisc,
				float4&			force)
{
	float coeff_max = 0.0f;
	for (uint i = 0; i < d_numplanes; ++i) {
		float coeff = PlaneForce(gridPos, pos, mass, d_plane[i], vel, dynvisc, force);
		if (coeff > coeff_max)
			coeff_max = coeff;
	}

	return coeff_max;
}

//! DOC-TODO describe function
__device__ __forceinline__ float
DemLJForce(	const texture<float, 2, hipReadModeElementType> texref,
			const int3&	gridPos,
			const float3&	pos,
			const float		mass,
			const float3&	vel,
			const float		dynvisc,
			float4&			force)
{
	const float2 demPos = DemPos(gridPos, pos);

	const float globalZ = d_worldOrigin.z + (gridPos.z + 0.5f)*d_cellSize.z + pos.z;
	const float globalZ0 = DemInterpol(texref, demPos);

	if (globalZ - globalZ0 < d_demzmin) {
		const plane_t demPlane(DemTangentPlane(texref, gridPos, pos, demPos, globalZ0));

		return PlaneForce(gridPos, pos, mass, demPlane, vel, dynvisc, force);
	}
	return 0;
}

/************************************************************************************************************/

/************************************************************************************************************/
/*		   Kernels for computing SPS tensor and SPS viscosity												*/
/************************************************************************************************************/

//! A functor that writes out turbvisc for SPS visc
template<bool>
struct write_sps_turbvisc
{
	template<typename FP>
	__device__ __forceinline__
	static void
	with(FP const& params, const uint index, const float turbvisc)
	{ /* do nothing */ }
};

template<>
template<typename FP>
__device__ __forceinline__ void
write_sps_turbvisc<true>::with(FP const& params, const uint index, const float turbvisc)
{ params.turbvisc[index] = turbvisc; }

//! A functor that writes out tau for SPS visc
template<bool>
struct write_sps_tau
{
	template<typename FP>
	__device__ __forceinline__
	static void
	with(FP const& params, const uint index, const float2& tau0, const float2& tau1, const float2& tau2)
	{ /* do nothing */ }
};

template<>
template<typename FP>
__device__ __forceinline__ void
write_sps_tau<true>::with(FP const& params, const uint index, const float2& tau0,
							const float2& tau1, const float2& tau2)
{
	params.tau0[index] = tau0;
	params.tau1[index] = tau1;
	params.tau2[index] = tau2;
}

/************************************************************************************************************/

/************************************************************************************************************/
/*		Device functions used in kernels other than the main forces kernel									*/
/************************************************************************************************************/

//! Computes boundary conditions at open boundaries
/*!
 Depending on whether velocity or pressure is prescribed at a boundary the respective other component
 is computed using the appropriate Riemann invariant.
*/
__device__ __forceinline__ void
calculateIOboundaryCondition(
			float4			&eulerVel,
	const	particleinfo	info,
	const	float			rhoInt,
	const	float			rhoExt,
	const	float3			uInt,
	const	float			unInt,
	const	float			unExt,
	const	float3			normal)
{
	const int a = fluid_num(info);
	const float rInt = R(rhoInt, a);

	// impose velocity (and k,eps) => compute density
	if (VEL_IO(info)) {
		float riemannR = 0.0f;
		if (unExt <= unInt) // Expansion wave
			riemannR = rInt + (unExt - unInt);
		else { // Shock wave
			float riemannRho = RHO(P(rhoInt, a) + rhoInt * unInt * (unInt - unExt), a);
			riemannR = R(riemannRho, a);
			float riemannC = soundSpeed(riemannRho, a);
			float lambda = unExt + riemannC;
			const float cInt = soundSpeed(rhoInt, a);
			float lambdaInt = unInt + cInt;
			if (lambda <= lambdaInt) // must be a contact discontinuity then (which would actually mean lambda == lambdaInt
				riemannR = rInt;
		}
		eulerVel.w = RHOR(riemannR, a);
	}
	// impose pressure => compute velocity (normal & tangential; k and eps are already interpolated)
	else {
		float flux = 0.0f;
		// Rankine-Hugoniot is not properly working
		const float cExt = soundSpeed(rhoExt, a);
		const float cInt = soundSpeed(rhoInt, a);
		const float lambdaInt = unInt + cInt;
		const float rExt = R(rhoExt, a);
		if (rhoExt <= rhoInt) { // Expansion wave
			flux = unInt + (rExt - rInt);
			float lambda = flux + cExt;
			if (lambda > lambdaInt) { // shock wave
				flux = (P(rhoInt, a) - P(rhoExt, a))/(rhoInt*fmax(unInt,1e-5f*d_sscoeff[a])) + unInt;
				// check that unInt was not too small
				if (fabs(flux) > d_sscoeff[a] * 0.1f)
					flux = unInt;
				lambda = flux + cExt;
				if (lambda <= lambdaInt) // contact discontinuity
					flux = unInt;
			}
		}
		else { // shock wave
			flux = (P(rhoInt, a) - P(rhoExt, a))/(rhoInt*fmax(unInt,1e-5f*d_sscoeff[a])) + unInt;
			// check that unInt was not too small
			if (fabs(flux) > d_sscoeff[a] * 0.1f)
				flux = unInt;
			float lambda = flux + cExt;
			if (lambda <= lambdaInt) { // expansion wave
				flux = unInt + (rExt - rInt);
				lambda = flux + cExt;
				if (lambda > lambdaInt) // contact discontinuity
					flux = unInt;
			}
		}
		// AM-TODO allow imposed tangential velocity (make sure normal component is zero)
		// currently for inflow we assume that the tangential velocity is zero
		as_float3(eulerVel) = make_float3(0.0f);
		// if the imposed pressure on the boundary is negative make sure that the flux is negative
		// as well (outflow)
		if (rhoExt < d_rho0[a])
			flux = fmin(flux, 0.0f);
		// Outflow
		if (flux < 0.0f)
			// impose eulerVel according to dv/dn = 0
			// and remove normal component of velocity
			as_float3(eulerVel) = uInt - dot(uInt, normal)*normal;
		// add calculated normal velocity
		as_float3(eulerVel) += normal*flux;
		// set density to the imposed one
		eulerVel.w = rhoExt;
	}
}

//! Determines the distribution of mass based on a position on a segment
/*!
 A position inside a segment is used to split the segment area into three parts. The respective
 size of these parts are used to determine how much the mass is redistributed that is associated
 with this position. This is used in two cases:

 1.) A mass flux is given or computed for a certain segment, then the position for the function
     is equivalent to the segement position. This determines the mass flux for the vertices

 2.) A fluid particle traverses a segment. Then the position is equal to the fluid position and
     the function determines how much mass of the fluid particle is distributed to each vertex
*/
__device__ __forceinline__ void
getMassRepartitionFactor(	const	float3	*vertexRelPos,
							const	float3	normal,
									float3	&beta)
{
	float3 v01 = vertexRelPos[0]-vertexRelPos[1];
	float3 v02 = vertexRelPos[0]-vertexRelPos[2];
	float3 p0  = vertexRelPos[0]-dot(vertexRelPos[0], normal)*normal;
	float3 p1  = vertexRelPos[1]-dot(vertexRelPos[1], normal)*normal;
	float3 p2  = vertexRelPos[2]-dot(vertexRelPos[2], normal)*normal;

	float refSurface = 0.5*dot(cross(v01, v02), normal);

	float3 v21 = vertexRelPos[2]-vertexRelPos[1];

	float surface0 = 0.5*dot(cross(p2, v21), normal);
	float surface1 = 0.5*dot(cross(p0, v02), normal);
	// Warning v10 = - v01
	float surface2 = - 0.5*dot(cross(p1, v01), normal);
	if (surface0 < 0. && surface2 < 0.) {
		// the projected point is clipped to v1
		surface0 = 0.;
		surface1 = refSurface;
		surface2 = 0.;
	} else if (surface0 < 0. && surface1 < 0.) {
		// the projected point is clipped to v2
		surface0 = 0.;
		surface1 = 0.;
		surface2 = refSurface;
	} else if (surface1 < 0. && surface2 < 0.) {
		// the projected point is clipped to v0
		surface0 = refSurface;
		surface1 = 0.;
		surface2 = 0.;
	} else if (surface0 < 0.) {
		// We project p2 into the v21 line, parallel to p0
		// then surface0 is 0
		// we also modify p0 an p1 accordingly
		float coef = surface0/(0.5*dot(cross(p0, v21), normal));

		p1 -= coef*p0;
		p0 *= (1.-coef);

		surface0 = 0.;
		surface1 = 0.5*dot(cross(p0, v02), normal);
		surface2 = - 0.5*dot(cross(p1, v01), normal);
	} else if (surface1 < 0.) {
		// We project p0 into the v02 line, parallel to p1
		// then surface1 is 0
		// we also modify p1 an p2 accordingly
		float coef = surface1/(0.5*dot(cross(p1, v02), normal));
		p2 -= coef*p1;
		p1 *= (1.-coef);

		surface0 = 0.5*dot(cross(p2, v21), normal);
		surface1 = 0.;
		surface2 = - 0.5*dot(cross(p1, v01), normal);
	} else if (surface2 < 0.) {
		// We project p1 into the v01 line, parallel to p2
		// then surface2 is 0
		// we also modify p0 an p2 accordingly
		float coef = -surface2/(0.5*dot(cross(p2, v01), normal));
		p0 -= coef*p2;
		p2 *= (1.-coef);

		surface0 = 0.5*dot(cross(p2, v21), normal);
		surface1 = 0.5*dot(cross(p0, v02), normal);
		surface2 = 0.;
	}

	beta.x = surface0/refSurface;
	beta.y = surface1/refSurface;
	beta.z = surface2/refSurface;
}

//! contribution of neighbor at relative position relPos with weight w to the MLS matrix mls
__device__ __forceinline__ void
MlsMatrixContrib(symtensor4 &mls, float4 const& relPos, float w)
{
	mls.xx += w;						// xx = ∑Wij*Vj
	mls.xy += relPos.x*w;				// xy = ∑(xi - xj)*Wij*Vj
	mls.xz += relPos.y*w;				// xz = ∑(yi - yj)*Wij*Vj
	mls.xw += relPos.z*w;				// xw = ∑(zi - zj)*Wij*Vj
	mls.yy += relPos.x*relPos.x*w;		// yy = ∑(xi - xj)^2*Wij*Vj
	mls.yz += relPos.x*relPos.y*w;		// yz = ∑(xi - xj)(yi - yj)*Wij*Vj
	mls.yw += relPos.x*relPos.z*w;		// yz = ∑(xi - xj)(zi - zj)*Wij*Vj
	mls.zz += relPos.y*relPos.y*w;		// zz = ∑(yi - yj)^2*Wij*Vj
	mls.zw += relPos.y*relPos.z*w;		// zz = ∑(yi - yj)(zi - zj)*Wij*Vj
	mls.ww += relPos.z*relPos.z*w;		// zz = ∑(yi - yj)^2*Wij*Vj

}

//! MLS contribution
/*!
 contribution of neighbor at relative position relPos with weight w to the
 MLS correction when B is the first row of the inverse MLS matrix
*/
__device__ __forceinline__ float
MlsCorrContrib(float4 const& B, float4 const& relPos, float w)
{
	return (B.x + B.y*relPos.x + B.z*relPos.y + B.w*relPos.z)*w;
	// ρ = ∑(ß0 + ß1(xi - xj) + ß2(yi - yj))*Wij*Vj
}

//! Fetch tau tensor from texture
/*!
 an auxiliary function that fetches the tau tensor
 for particle i from the textures where it's stored
*/
__device__
symtensor3 fetchTau(uint i)
{
	symtensor3 tau;
	float2 temp = tex1Dfetch(tau0Tex, i);
	tau.xx = temp.x;
	tau.xy = temp.y;
	temp = tex1Dfetch(tau1Tex, i);
	tau.xz = temp.x;
	tau.yy = temp.y;
	temp = tex1Dfetch(tau2Tex, i);
	tau.yz = temp.x;
	tau.zz = temp.y;
	return tau;
}

/*  @} */

/** \name Kernels
 *  @{ */

//! Compute SPS matrix
/*!
 Compute the Sub-Particle-Stress (SPS) Tensor matrix for all Particles
 WITHOUT Kernel correction

 Procedure:

 (1) compute velocity gradients

 (2) compute turbulent eddy viscosity (non-dynamic)

 (3) compute turbulent shear stresses

 (4) return SPS tensor matrix (tau) divided by rho^2
*/
template<KernelType kerneltype,
	BoundaryType boundarytype,
	uint simflags>
__global__ void
__launch_bounds__(BLOCK_SIZE_SPS, MIN_BLOCKS_SPS)
SPSstressMatrixDevice(sps_params<kerneltype, boundarytype, simflags> params)
{
	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;

	if (index >= params.numParticles)
		return;

	// read particle data from sorted arrays
	// Compute SPS matrix only for any kind of particles
	// TODO testpoints should also compute SPS, it'd be useful
	// when we will enable SPS saving to disk
	const particleinfo info = tex1Dfetch(infoTex, index);

	// read particle data from sorted arrays
	#if( __COMPUTE__ >= 20)
	const float4 pos = params.pos[index];
	#else
	const float4 pos = tex1Dfetch(posTex, index);
	#endif

	// skip inactive particles
	if (INACTIVE(pos))
		return;

	const float4 vel = tex1Dfetch(velTex, index);

	// Gradients of the the velocity components
	float3 dvx = make_float3(0.0f);
	float3 dvy = make_float3(0.0f);
	float3 dvz = make_float3(0.0f);

	// Compute grid position of current particle
	const int3 gridPos = calcGridPosFromParticleHash( params.particleHash[index] );

	// Persistent variables across getNeibData calls
	char neib_cellnum = -1;
	uint neib_cell_base_index = 0;
	float3 pos_corr;

	// loop over all the neighbors
	for (idx_t i = 0; i < d_neiblist_end; i += d_neiblist_stride) {
		neibdata neib_data = params.neibsList[i + index];

		if (neib_data == 0xffff) break;

		const uint neib_index = getNeibIndex(pos, pos_corr, params.cellStart,
				neib_data, gridPos, neib_cellnum, neib_cell_base_index);

		// Compute relative position vector and distance
		// Now relPos is a float4 and neib mass is stored in relPos.w
		#if( __COMPUTE__ >= 20)
		const float4 relPos = pos_corr - params.pos[neib_index];
		#else
		const float4 relPos = pos_corr - tex1Dfetch(posTex, neib_index);
		#endif

		// skip inactive particles
		if (INACTIVE(relPos))
			continue;

		const float r = length3(relPos);

		// Compute relative velocity
		// Now relVel is a float4 and neib density is stored in relVel.w
		const float4 relVel = as_float3(vel) - tex1Dfetch(velTex, neib_index);
		const particleinfo neib_info = tex1Dfetch(infoTex, neib_index);

		// Velocity gradient is contributed by all particles
		// TODO: fix SA case
		if ( r < params.influenceradius ) {
			const float f = F<kerneltype>(r, params.slength)*relPos.w/relVel.w;	// 1/r ∂Wij/∂r Vj

			// Velocity Gradients
			dvx -= relVel.x*as_float3(relPos)*f;	// dvx = -∑mj/ρj vxij (ri - rj)/r ∂Wij/∂r
			dvy -= relVel.y*as_float3(relPos)*f;	// dvy = -∑mj/ρj vyij (ri - rj)/r ∂Wij/∂r
			dvz -= relVel.z*as_float3(relPos)*f;	// dvz = -∑mj/ρj vzij (ri - rj)/r ∂Wij/∂r
			}
		} // end of loop through neighbors


	// SPS stress matrix elements
	symtensor3 tau;

	// Calculate Sub-Particle Scale viscosity
	// and special turbulent terms
	float SijSij_bytwo = 2.0f*(dvx.x*dvx.x + dvy.y*dvy.y + dvz.z*dvz.z);	// 2*SijSij = 2.0((∂vx/∂x)^2 + (∂vy/∂yx)^2 + (∂vz/∂z)^2)
	float temp = dvx.y + dvy.x;		// 2*SijSij += (∂vx/∂y + ∂vy/∂x)^2
	tau.xy = temp;
	SijSij_bytwo += temp*temp;
	temp = dvx.z + dvz.x;			// 2*SijSij += (∂vx/∂z + ∂vz/∂x)^2
	tau.xz = temp;
	SijSij_bytwo += temp*temp;
	temp = dvy.z + dvz.y;			// 2*SijSij += (∂vy/∂z + ∂vz/∂y)^2
	tau.yz = temp;
	SijSij_bytwo += temp*temp;
	const float S = sqrtf(SijSij_bytwo);
	const float nu_SPS = d_smagfactor*S;		// Dalrymple & Rogers (2006): eq. (12)
	const float divu_SPS = 0.6666666666f*nu_SPS*(dvx.x + dvy.y + dvz.z);
	const float Blinetal_SPS = d_kspsfactor*SijSij_bytwo;

	// Storing the turbulent viscosity for each particle
	write_sps_turbvisc<simflags & SPSK_STORE_TURBVISC>::with(params, index, nu_SPS);

	// Shear Stress matrix = TAU (pronounced taf)
	// Dalrymple & Rogers (2006): eq. (10)
	if (simflags & SPSK_STORE_TAU) {

		tau.xx = nu_SPS*(dvx.x + dvx.x) - divu_SPS - Blinetal_SPS;	// tau11 = tau_xx/ρ^2
		tau.xx /= vel.w;
		tau.xy *= nu_SPS/vel.w;								// tau12 = tau_xy/ρ^2
		tau.xz *= nu_SPS/vel.w;								// tau13 = tau_xz/ρ^2
		tau.yy = nu_SPS*(dvy.y + dvy.y) - divu_SPS - Blinetal_SPS;	// tau22 = tau_yy/ρ^2
		tau.yy /= vel.w;
		tau.yz *= nu_SPS/vel.w;								// tau23 = tau_yz/ρ^2
		tau.zz = nu_SPS*(dvz.z + dvz.z) - divu_SPS - Blinetal_SPS;	// tau33 = tau_zz/ρ^2
		tau.zz /= vel.w;

		write_sps_tau<simflags & SPSK_STORE_TAU>::with(params, index, make_float2(tau.xx, tau.xy),
				make_float2(tau.xz, tau.yy), make_float2(tau.yz, tau.zz));
	}
}
/************************************************************************************************************/

/************************************************************************************************************/
/*										Density computation							*/
/************************************************************************************************************/

//! Continuity equation with the Grenier formulation
/*!
 When using the Grenier formulation, density is reinitialized at each timestep from
 a Shepard-corrected mass distribution limited to same-fluid particles M and volumes ω computed
 from a continuity equation, with ρ = M/ω.
 During the same run, we also compute σ, the discrete specific volume
 (see e.g. Hu & Adams 2005), obtained by summing the kernel computed over
 _all_ neighbors (not just the same-fluid ones) which is used in the continuity
 equation as well as the Navier-Stokes equation
*/
template<KernelType kerneltype, BoundaryType boundarytype>
__global__ void
densityGrenierDevice(
			float* __restrict__		sigmaArray,
	const	float4* __restrict__	posArray,
			float4* __restrict__	velArray,
	const	particleinfo* __restrict__	infoArray,
	const	hashKey* __restrict__	particleHash,
	const	float4* __restrict__	volArray,
	const	uint* __restrict__		cellStart,
	const	neibdata* __restrict__	neibsList,
	const	uint	numParticles,
	const	float	slength,
	const	float	influenceradius)
{
	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;

	if (index >= numParticles)
		return;

	const particleinfo info = infoArray[index];

	/* We only process FLUID particles normally,
	   except with DYN_BOUNDARY, where we also process boundary particles
	   */
	if (boundarytype != DYN_BOUNDARY && NOT_FLUID(info))
		return;

	const float4 pos = posArray[index];

	if (INACTIVE(pos))
		return;

	const ushort fnum = fluid_num(info);
	const float vol = volArray[index].w;
	float4 vel = velArray[index];

	// self contribution
	float corr = W<kerneltype>(0, slength);
	float sigma = corr;
	float mass_corr = pos.w*corr;

	const int3 gridPos = calcGridPosFromParticleHash( particleHash[index] );
	// Persistent variables across getNeibData calls
	char neib_cellnum = 0;
	uint neib_cell_base_index = 0;
	float3 pos_corr;

	// For DYN_BOUNDARY particles, we compute sigma in the same way as fluid particles,
	// except that if the boundary particle has no fluid neighbors we set its
	// sigma to a default value which is the 'typical' specific volume, given by
	// the typical number of neighbors divided by the volume of the influence sphere
	bool has_fluid_neibs = false;

	// Loop over all neighbors
	for (idx_t i = 0; i < d_neiblist_end; i += d_neiblist_stride) {
		neibdata neib_data = neibsList[i + index];

		if (neib_data == 0xffff) break;

		const uint neib_index = getNeibIndex(pos, pos_corr, cellStart, neib_data, gridPos,
			neib_cellnum, neib_cell_base_index);

		// Compute relative position vector and distance

		const particleinfo neib_info = infoArray[neib_index];
		const float4 relPos = pos_corr - posArray[neib_index];
		float r = length(as_float3(relPos));

		/* Contributions only come from active particles within the influence radius
		   that are fluid particles (or also non-fluid in DYN_BOUNDARY case).
		   TODO check what to do with SA
		   Sigma calculations uses all such particles, whereas smoothed mass
		   only uses same-fluid particles.
		   Note that this requires PT_BOUNDARY neighbors to be in the list for
		   PT_BOUNDARY particles, lest the boundary particles end up assuming
		   they are always on the free surface.
		   TODO an alternative approach for DYN_BOUNDARY would be to assign
		   the sigma from the closest fluid particle, but that would require
		   two runs, one for fluid and one for neighbor particles.
		 */
		if (INACTIVE(relPos) || r >= influenceradius ||
			((boundarytype != DYN_BOUNDARY) && NOT_FLUID(neib_info)))
			continue;

		const float w = W<kerneltype>(r, slength);
		sigma += w;
		if (FLUID(neib_info))
			has_fluid_neibs = true;

		/* For smoothed mass, fluid particles only consider fluid particles,
		   and non-fluid (only present for DYN_BOUNDARY) only consider non-fluid
		   */
		if ((boundarytype != DYN_BOUNDARY || (PART_TYPE(neib_info) == PART_TYPE(info)))
			&& fluid_num(neib_info) == fnum) {
			mass_corr += relPos.w*w;
			corr += w;
		}
	}

	if (boundarytype == DYN_BOUNDARY && NOT_FLUID(info) && !has_fluid_neibs) {
		// TODO OPTIMIZE
		const float typical_sigma = 3*cuneibs::d_maxNeibs/
			(4*M_PIf*influenceradius*influenceradius*influenceradius);
		sigma = typical_sigma;
	}

	// M = mass_corr/corr, ρ = M/ω
	// this could be optimized to pos.w/vol assuming all same-fluid particles
	// have the same mass
	vel.w = mass_corr/(corr*vol);
	velArray[index] = vel;
	sigmaArray[index] = sigma;
}

/************************************************************************************************************/

// flags for the vertexinfo .w coordinate which specifies how many vertex particles of one segment
// is associated to an open boundary
#define VERTEX1 ((flag_t)1)
#define VERTEX2 (VERTEX1 << 1)
#define VERTEX3 (VERTEX2 << 1)
#define ALLVERTICES ((flag_t)(VERTEX1 | VERTEX2 | VERTEX3))

//! Computes the boundary condition on segments for SA boundaries
/*!
 This function computes the boundary condition for density/pressure on segments if the SA boundary type
 is selected. It does this not only for solid wall boundaries but also open boundaries. Additionally,
 this function detects when a fluid particle crosses the open boundary and it identifies which segment it
 crossed. The vertices of this segment are then used to identify how the mass of this fluid particle is
 split.
*/
template<KernelType kerneltype>
__global__ void
__launch_bounds__(BLOCK_SIZE_SHEPARD, MIN_BLOCKS_SHEPARD)
saSegmentBoundaryConditions(			float4*		oldPos,
										float4*		oldVel,
										float*		oldTKE,
										float*		oldEps,
										float4*		oldEulerVel,
										float4*		oldGGam,
										vertexinfo*	vertices,
								const	uint*		vertIDToIndex,
								const	float2*		vertPos0,
								const	float2*		vertPos1,
								const	float2*		vertPos2,
								const	hashKey*	particleHash,
								const	uint*		cellStart,
								const	neibdata*	neibsList,
								const	uint		numParticles,
								const	float		deltap,
								const	float		slength,
								const	float		influenceradius,
								const	bool		initStep,
								const	uint		step,
								const	bool		inoutBoundaries)
{
	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;

	if (index >= numParticles)
		return;

	// read particle data from sorted arrays
	const particleinfo info = tex1Dfetch(infoTex, index);

	// For boundary segments this kernel computes the boundary conditions
	if (BOUNDARY(info)) {

		float4 eulerVel = make_float4(0.0f);
		float tke = 0.0f;
		float eps = 0.0f;
		const vertexinfo verts = vertices[index];

		// load the indices of the vertices only once
		const uint vertXidx = vertIDToIndex[verts.x];
		const uint vertYidx = vertIDToIndex[verts.y];
		const uint vertZidx = vertIDToIndex[verts.z];

		// get the imposed quantities from the arrays which were set in the problem specific routines
		if (IO_BOUNDARY(info)) {
			// for imposed velocity the velocity, tke and eps are required and only rho will be calculated
			if (VEL_IO(info)) {
				eulerVel = oldEulerVel[index];
				eulerVel.w = 0.0f;
				if (oldTKE)
					tke = oldTKE[index];
				if (oldEps)
					eps = oldEps[index];
			}
			// for imposed density only eulerVel.w will be required, the rest will be computed
			else
				eulerVel.w = oldEulerVel[index].w;
		}

		// velocity for moving objects transferred from vertices
		float3 vel = make_float3(0.0f);
		if (MOVING(info)) {
			vel += as_float3(oldVel[vertXidx]);
			vel += as_float3(oldVel[vertYidx]);
			vel += as_float3(oldVel[vertZidx]);
			vel /= 3.0f;
		}
		as_float3(oldVel[index]) = vel;

		// compute an average gamma for the segment
		float gam = oldGGam[index].w;
		if (gam < 1e-10) {
			float4 gGam = (oldGGam[vertXidx] + oldGGam[vertYidx] + oldGGam[vertZidx])/3.0f;
			oldGGam[index] = gGam;
			gam = gGam.w;
		}

		const float4 pos = oldPos[index];

		// note that all sums below run only over fluid particles (including the Shepard filter)
		float sumpWall = 0.0f; // summation for computing the density
		float sump = 0.0f; // summation for computing the pressure
		float3 sumvel = make_float3(0.0f); // summation to compute the internal velocity for open boundaries
		float sumtke = 0.0f; // summation for computing tke (k-epsilon model)
		float sumeps = 0.0f; // summation for computing epsilon (k-epsilon model)
		float alpha  = 0.0f;  // the shepard filter

		// Compute grid position of current particle
		const int3 gridPos = calcGridPosFromParticleHash( particleHash[index] );

		// Persistent variables across getNeibData calls
		char neib_cellnum = 0;
		uint neib_cell_base_index = 0;
		float3 pos_corr;

		// Square of sound speed. Would need modification for multifluid
		const float sqC0 = d_sqC0[fluid_num(info)];

		const float4 normal = tex1Dfetch(boundTex, index);

		// Loop over all the neighbors
		for (idx_t i = 0; i < d_neiblist_end; i += d_neiblist_stride) {
			neibdata neib_data = neibsList[i + index];

			if (neib_data == 0xffff) break;

			const uint neib_index = getNeibIndex(pos, pos_corr, cellStart, neib_data, gridPos,
						neib_cellnum, neib_cell_base_index);

			// Compute relative position vector and distance
			// Now relPos is a float4 and neib mass is stored in relPos.w
			const float4 relPos = pos_corr - oldPos[neib_index];

			// skip inactive particles
			if (INACTIVE(relPos))
				continue;

			const float r = length(as_float3(relPos));
			const particleinfo neib_info = tex1Dfetch(infoTex, neib_index);

			if (dot3(normal, relPos) < 0.0f &&
				r < influenceradius &&
				FLUID(neib_info)
				//(FLUID(neib_info) || (!IO_BOUNDARY(info) && VERTEX(neib_info) && IO_BOUNDARY(neib_info) && !CORNER(neib_info)))
				//(FLUID(neib_info) || (VERTEX(neib_info) && !IO_BOUNDARY(neib_info) && IO_BOUNDARY(info)))
			   ){
				const float neib_rho = oldVel[neib_index].w;

				const float neib_pres = P(neib_rho, fluid_num(neib_info));
				const float neib_vel = length(make_float3(oldVel[neib_index]));
				const float neib_k = oldTKE ? oldTKE[neib_index] : NAN;
				const float neib_eps = oldEps ? oldEps[neib_index] : NAN;

				// kernel value times volume
				const float w = W<kerneltype>(r, slength)*relPos.w/neib_rho;
				// normal distance based on grad Gamma which approximates the normal of the domain
				const float normDist = fmax(fabs(dot3(normal,relPos)), deltap);
				sumpWall += fmax(neib_pres + neib_rho*dot(d_gravity, as_float3(relPos)), 0.0f)*w;
				// for all boundaries we have dk/dn = 0
				sumtke += w*neib_k;
				if (IO_BOUNDARY(info)) {
					sumvel += w*as_float3(oldVel[neib_index] + oldEulerVel[neib_index]);
					// for open boundaries compute pressure interior state
					//sump += w*fmax(0.0f, neib_pres+dot(d_gravity, as_float3(relPos)*d_rho0[fluid_num(neib_info)]));
					sump += w*fmax(0.0f, neib_pres);
					// and de/dn = 0
					sumeps += w*neib_eps;
				}
				else
					// for solid boundaries we have de/dn = c_mu^(3/4)*4*k^(3/2)/(\kappa r)
					// the constant is coming from 4*powf(0.09,0.75)/0.41
					sumeps += w*(neib_eps + 1.603090412f*powf(neib_k,1.5f)/normDist);
				alpha += w;
			}
		}

		if (IO_BOUNDARY(info)) {
			if (alpha > 0.1f*gam) {
				sumvel /= alpha;
				sump /= alpha;
				oldVel[index].w = RHO(sump, fluid_num(info));
				if (VEL_IO(info)) {
					// for velocity imposed boundaries we impose k and epsilon
					if (oldTKE)
						oldTKE[index] = tke;
					if (oldEps)
						oldEps[index] = eps;
				}
				else {
					oldEulerVel[index] = make_float4(0.0f);
					// for pressure imposed boundaries we take dk/dn = 0
					if (oldTKE)
						oldTKE[index] = sumtke/alpha;
					// for pressure imposed boundaries we have de/dn = 0
					if (oldEps)
						oldEps[index] = sumeps/alpha;
				}
			}
			else {
				sump = 0.0f;
				if (VEL_IO(info)) {
					sumvel = as_float3(eulerVel);
					oldVel[index].w = d_rho0[fluid_num(info)];
				}
				else {
					sumvel = make_float3(0.0f);
					oldVel[index].w = oldEulerVel[index].w;
					oldEulerVel[index] = make_float4(0.0f, 0.0f, 0.0f, oldEulerVel[index].w);
				}
				if (oldTKE)
					oldTKE[index] = 1e-6f;
				if (oldEps)
					oldEps[index] = 1e-6f;
			}

			// compute Riemann invariants for open boundaries
			const float unInt = dot(sumvel, as_float3(normal));
			const float unExt = dot3(eulerVel, normal);
			const float rhoInt = oldVel[index].w;
			const float rhoExt = eulerVel.w;

			calculateIOboundaryCondition(eulerVel, info, rhoInt, rhoExt, sumvel, unInt, unExt, as_float3(normal));

			oldEulerVel[index] = eulerVel;
			// the density of the particle is equal to the "eulerian density"
			oldVel[index].w = eulerVel.w;

		}
		// non-open boundaries
		else {
			alpha = fmax(alpha, 0.1f*gam); // avoid division by 0
			// density condition
			oldVel[index].w = RHO(sumpWall/alpha,fluid_num(info));
			// k-epsilon boundary conditions
			if (oldTKE) {
				// k condition
				oldTKE[index] = sumtke/alpha;
				// eulerian velocity on the wall
				eulerVel = (	oldEulerVel[vertXidx] +
								oldEulerVel[vertYidx] +
								oldEulerVel[vertZidx] )/3.0f;
				// ensure that velocity is normal to segment normal
				eulerVel -= dot3(eulerVel,normal)*normal;
				oldEulerVel[index] = eulerVel;
			}
			// if k-epsilon is not used but oldEulerVel is present (for open boundaries) set it to 0
			else if (oldEulerVel)
				oldEulerVel[index] = make_float4(0.0f);
			// epsilon condition
			if (oldEps)
				// for solid boundaries we have de/dn = 4 0.09^0.075 k^1.5/(0.41 r)
				oldEps[index] = fmax(sumeps/alpha,1e-5f); // eps should never be 0
		}

	}
	// for fluid particles this kernel checks whether they have crossed the boundary at open boundaries
	else if (inoutBoundaries && step==2 && FLUID(info)) {

		float4 pos = oldPos[index];

		// don't check inactive particles and those that have already found their segment
		if (INACTIVE(pos) || vertices[index].x | vertices[index].y != 0)
			return;

		// Compute grid position of current particle
		const int3 gridPos = calcGridPosFromParticleHash( particleHash[index] );

		// Persistent variables across getNeibData calls
		char neib_cellnum = 0;
		uint neib_cell_base_index = 0;
		float3 pos_corr;

		const float4 vel = oldVel[index];

		float rSqMin = influenceradius*influenceradius;
		uint neib_indexMin = UINT_MAX;
		float4 relPosMin = make_float4(0.0f);

		// Loop over all the neighbors
		for (idx_t i = 0; i < d_neiblist_end; i += d_neiblist_stride) {
			neibdata neib_data = neibsList[i + index];

			if (neib_data == 0xffff) break;

			const uint neib_index = getNeibIndex(pos, pos_corr, cellStart, neib_data, gridPos,
						neib_cellnum, neib_cell_base_index);
			const particleinfo neib_info = tex1Dfetch(infoTex, neib_index);

			// for open boundary segments check whether this fluid particle has crossed the boundary
			// in order to do so we try to identify the closest segment which the particle has passed
			if (BOUNDARY(neib_info) && IO_BOUNDARY(neib_info)) {

				// Compute relative position vector and distance
				// Now relPos is a float4 and neib mass is stored in relPos.w
				const float4 relPos = pos_corr - oldPos[neib_index];

				const float4 normal = tex1Dfetch(boundTex, neib_index);

				const float3 relVel = as_float3(vel - oldVel[neib_index]);

				const float rSq = sqlength3(relPos);

				// check if we are behind a segment
				// additionally check if the velocity vector is pointing outwards
				if (dot3(normal, relPos) <= 0.0f &&
					rSq < rSqMin &&
					dot(relVel, as_float3(normal)) < 0.0f) {
					// this can only be reached if the segment is closer than all those before, so we save its distance
					rSqMin = rSq;
					// its relative position
					relPosMin = relPos;
					// and also its index
					neib_indexMin = neib_index;
				}
			}
		} // end neighbour loop

		// if we have found a segment that was crossed and that is close by
		if (neib_indexMin != UINT_MAX) {
			const float4 normal = tex1Dfetch(boundTex, neib_indexMin);
			// first get the position of the vertices local coordinate system for relative positions to vertices
			uint j = 0;
			// Get index j for which n_s is minimal
			if (fabs(normal.x) > fabs(normal.y))
				j = 1;
			if ((1-j)*fabs(normal.x) + j*fabs(normal.y) > fabs(normal.z))
				j = 2;

			// compute the first coordinate which is a 2-D rotated version of the normal
			const float4 coord1 = normalize(make_float4(
				// switch over j to give: 0 -> (0, z, -y); 1 -> (-z, 0, x); 2 -> (y, -x, 0)
				-((j==1)*normal.z) +  (j == 2)*normal.y , // -z if j == 1, y if j == 2
				  (j==0)*normal.z  - ((j == 2)*normal.x), // z if j == 0, -x if j == 2
				-((j==0)*normal.y) +  (j == 1)*normal.x , // -y if j == 0, x if j == 1
				0));
			// the second coordinate is the cross product between the normal and the first coordinate
			const float4 coord2 = cross3(normal, coord1);

			const float2 vPos0 = vertPos0[neib_indexMin];
			const float2 vPos1 = vertPos1[neib_indexMin];
			const float2 vPos2 = vertPos2[neib_indexMin];

			// relative positions of vertices with respect to the segment, normalized by h
			float4 v0 = -(vPos0.x*coord1 + vPos0.y*coord2); // e.g. v0 = r_{v0} - r_s
			float4 v1 = -(vPos1.x*coord1 + vPos1.y*coord2);
			float4 v2 = -(vPos2.x*coord1 + vPos2.y*coord2);

			// the fluid particle found a segment so let's save it
			// note normally vertices is empty for fluid particles so this will indicate
			// from now on that it has to be destroyed
			vertexinfo verts = vertices[neib_indexMin];

			// furthermore we need to save the weights beta_{a,v} to avoid using
			// neighbours of neighbours. As the particle will be deleted anyways we
			// just use the velocity array which we don't need anymore. The beta_{a,v}
			// in the 3-D case are based on surface areas based on the triangle partition
			// governed by the position of the fluid particle
			float4 vertexWeights = make_float4(0.0f);
			const float3 vx[3] = {as_float3(relPosMin - v0), as_float3(relPosMin - v1), as_float3(relPosMin - v2)};
			getMassRepartitionFactor(vx, as_float3(normal), as_float3(vertexWeights));
			// transfer mass to .w index as it is overwritten with the disable below
			vertexWeights.w = pos.w;
			oldGGam[index] = vertexWeights;
			vertices[index] = verts;
		}
	}
}

/// Compute boundary conditions for vertex particles in the semi-analytical boundary case
/*! This function determines the physical properties of vertex particles in the semi-analytical boundary case. The properties of fluid particles are used to compute the properties of the vertices. Due to this most arrays are read from (the fluid info) and written to (the vertex info) simultaneously inside this function. In the case of open boundaries the vertex mass is updated in this routine and new fluid particles are created on demand. Additionally, the mass of outgoing fluid particles is redistributed to vertex particles herein.
 *	\param[in,out] oldPos : pointer to positions and masses; masses of vertex particles are updated
 *	\param[in,out] oldVel : pointer to velocities and density; densities of vertex particles are updated
 *	\param[in,out] oldTKE : pointer to turbulent kinetic energy
 *	\param[in,out] oldEps : pointer to turbulent dissipation
 *	\param[in,out] oldGGam : pointer to (grad) gamma; used only for cloning (i.e. creating a new particle)
 *	\param[in,out] oldEulerVel : pointer to Eulerian velocity & density; imposed values are set and the other is computed here
 *	\param[in,out] forces : pointer to forces; used only for cloning
 *	\param[in,out] contupd : pointer to contudp; used only for cloning
 *	\param[in,out] vertices : pointer to associated vertices; fluid particles have this information if they are passing through a boundary and are going to be deleted
 *	\param[in] vertIDToIndex : pointer that associated a vertex id with an array index
 *	\param[in] vertPos[0] : relative position of the vertex 0 with respect to the segment center
 *	\param[in] vertPos[1] : relative position of the vertex 1 with respect to the segment center
 *	\param[in] vertPos[2] : relative position of the vertex 2 with respect to the segment center
 *	\param[in,out] pinfo : pointer to particle info; written only when cloning
 *	\param[in,out] particleHash : pointer to particle hash; written only when cloning
 *	\param[in] cellStart : pointer to indices of first particle in cells
 *	\param[in] neibsList : neighbour list
 *	\param[in] numParticles : number of particles
 *	\param[out] newNumParticles : number of particles after creation of new fluid particles due to open boundaries
 *	\param[in] dt : time-step size
 *	\param[in] step : the step in the time integrator
 *	\param[in] deltap : the particle size
 *	\param[in] slength : the smoothing length
 *	\param[in] influenceradius : the kernel radius
 *	\param[in] deviceId : current device identifier
 *	\param[in] numDevices : total number of devices; used for id generation of new fluid particles
 */
template<KernelType kerneltype>
__global__ void
__launch_bounds__(BLOCK_SIZE_SHEPARD, MIN_BLOCKS_SHEPARD)
saVertexBoundaryConditions(
						float4*			oldPos,
						float4*			oldVel,
						float*			oldTKE,
						float*			oldEps,
						float4*			oldGGam,
						float4*			oldEulerVel,
						float4*			forces,
						float2*			contupd,
						vertexinfo*		vertices,
				const	float2*			vertPos0,
				const	float2*			vertPos1,
				const	float2*			vertPos2,
				const	uint*			vertIDToIndex,
						particleinfo*	pinfo,
						hashKey*		particleHash,
				const	uint*			cellStart,
				const	neibdata*		neibsList,
				const	uint			numParticles,
						uint*			newNumParticles,
				const	float			dt,
				const	int				step,
				const	float			deltap,
				const	float			slength,
				const	float			influenceradius,
				const	bool			initStep,
				const	bool			resume,
				const	uint			deviceId,
				const	uint			numDevices)
{
	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;

	if (index >= numParticles)
		return;

	// read particle data from sorted arrays
	// kernel is only run for vertex particles
	const particleinfo info = pinfo[index];
	if (!VERTEX(info))
		return;

	float4 pos = oldPos[index];

	const float vel = length(make_float3(oldVel[index]));

	// these are taken as the sum over all adjacent segments
	float sumpWall = 0.0f; // summation for computing the density
	float sumtke = 0.0f; // summation for computing tke (k-epsilon model)
	float sumeps = 0.0f; // summation for computing epsilon (k-epsilon model)
	float sumMdot = 0.0f; // summation for computing the mass variance based on in/outflow
	float massFluid = 0.0f; // mass obtained from a outgoing - mass of a new fluid
	float sump = 0.0f; // summation for the pressure on IO boundaries
	float3 sumvel = make_float3(0.0f); // summation for the velocity on IO boundaries
	float alpha = 0.0f; // summation of normalization for IO boundaries
	bool foundFluid = false; // check if a vertex particle has a fluid particle in its support
	float numseg = 0.0f;

	// Compute grid position of current particle
	const int3 gridPos = calcGridPosFromParticleHash( particleHash[index] );

	// Persistent variables across getNeibData calls
	char neib_cellnum = 0;
	uint neib_cell_base_index = 0;
	float3 pos_corr;
	const float gam = oldGGam[index].w;
	// normal:
	// for solid walls this normal only takes the associated normals of segments into account that are solid as well
	// for io walls this normal only takes the associated normals of segments into account that themeselves are io
	const float3 normal = as_float3(tex1Dfetch(boundTex, index));
	// wall normal:
	// for corner vertices the wall normal is equal to the normal of the associated segments that belong to a solid wall
	// at the initialization step the wall normal is computed for all vertices in order to get an approximate normal
	// which is then used to compute grad gamma and gamma
	float3 wallNormal = make_float3(0.0f);
	const float sqC0 = d_sqC0[fluid_num(info)];

	// Loop over all the neighbors
	for (idx_t i = 0; i < d_neiblist_end; i += d_neiblist_stride) {
		neibdata neib_data = neibsList[i + index];

		if (neib_data == 0xffff) break;

		const uint neib_index = getNeibIndex(pos, pos_corr, cellStart, neib_data, gridPos,
					neib_cellnum, neib_cell_base_index);

		const particleinfo neib_info = pinfo[neib_index];

		if (BOUNDARY(neib_info) || FLUID(neib_info)) {

			// prepare indices of neib vertices
			const vertexinfo neibVerts = vertices[neib_index];

			// load the indices of the vertices
			const uint neibVertXidx = vertIDToIndex[neibVerts.x];
			const uint neibVertYidx = vertIDToIndex[neibVerts.y];
			const uint neibVertZidx = vertIDToIndex[neibVerts.z];

			if (FLUID(neib_info)) {
			//if (FLUID(neib_info) || (VERTEX(neib_info) && !IO_BOUNDARY(neib_info) && IO_BOUNDARY(info))) {
			//if (FLUID(neib_info) || (!IO_BOUNDARY(info) && VERTEX(neib_info) && IO_BOUNDARY(neib_info) && !CORNER(neib_info))) {
				const float4 relPos = pos_corr - oldPos[neib_index];
				//if (INACTIVE(relPos) || dot(normal, as_float3(relPos)) > 0.0f)
				if (INACTIVE(relPos))
					continue;
				const float r = length(as_float3(relPos));

				if (r < influenceradius){
					const float neib_rho = oldVel[neib_index].w;
					const float neib_pres = P(neib_rho, fluid_num(neib_info));
					const float neib_vel = length(make_float3(oldVel[neib_index]));

					// kernel value times volume
					const float w = W<kerneltype>(r, slength)*relPos.w/neib_rho;
					// normal distance based on grad Gamma which approximates the normal of the domain
					sumpWall += fmax(neib_pres + neib_rho*dot(d_gravity, as_float3(relPos)), 0.0f)*w;
					// for all boundaries we have dk/dn = 0
					if (IO_BOUNDARY(info) && !CORNER(info)) {
						// for open boundaries compute dv/dn = 0
						sumvel += w*as_float3(oldVel[neib_index] + oldEulerVel[neib_index]);
						// for open boundaries compute pressure interior state
						//sump += w*fmax(0.0f, neib_pres+dot(d_gravity, as_float3(relPos)*d_rho0[fluid_num(neib_info)]));
						sump += w*fmax(0.0f, neib_pres);
						// and de/dn = 0
					}
					alpha += w;
				}
			}

			if (BOUNDARY(neib_info)) {
				const float4 boundElement = tex1Dfetch(boundTex, neib_index);

				// check if vertex is associated with this segment
				if (neibVertXidx == index || neibVertYidx == index || neibVertZidx == index) {
					// in the initial step we need to compute an approximate grad gamma direction
					// for the computation of gamma, in general we need a sort of normal as well
					// for corner vertices this wallNormal takes only solid walls into account so
					// that the eulerian velocity in the k-eps case is only normal to the solid wall
					if (initStep || (CORNER(info) && !IO_BOUNDARY(neib_info)))
						wallNormal += as_float3(boundElement)*boundElement.w;
					// k and eps are taken directly from the associated segments
					const float neib_k = oldTKE ? oldTKE[neib_index] : NAN;
					const float neib_eps = oldEps ? oldEps[neib_index] : NAN;
					sumtke += neib_k;
					sumeps += neib_eps;
					numseg += 1.0f;
					// corner vertices only take solid wall segments into account
					if (CORNER(info) && IO_BOUNDARY(neib_info))
						continue;
					// boundary conditions on rho, k, eps
					const float neibRho = oldVel[neib_index].w;
					if (!CORNER(info) && IO_BOUNDARY(neib_info)){
						/* The following would increase the output of particles close to an edge
						 * But it is not used for the following reason: If only 1/3 of each segment is taken into account
						 * it lowers the effective inflow area. This is ok, as part of the area of a segment that is associated
						 * with a corner "belongs" to a corner vertex.
						// number of vertices associated to a segment that are of the same object type
						float numOutVerts = 2.0f;
						if (neibVerts.w == ALLVERTICES) // all vertices are of the same object type
							numOutVerts = 3.0f;
						else if (neibVerts.w & ~VERTEX1 == 0 || neibVerts.w & ~VERTEX2 == 0 || neibVerts.w & ~VERTEX3 == 0) // only one vertex
							numOutVerts = 1.0f;
						*/
						/*
						// Distribute mass flux evenly among vertex particles of a segment
						float numOutVerts = 3.0f;
						*/

						// first get the position of the vertices local coordinate system for relative positions to vertices
						uint j = 0;
						// Get index j for which n_s is minimal
						if (fabs(boundElement.x) > fabs(boundElement.y))
							j = 1;
						if ((1-j)*fabs(boundElement.x) + j*fabs(boundElement.y) > fabs(boundElement.z))
							j = 2;

						// compute the first coordinate which is a 2-D rotated version of the normal
						const float4 coord1 = normalize(make_float4(
							// switch over j to give: 0 -> (0, z, -y); 1 -> (-z, 0, x); 2 -> (y, -x, 0)
							-((j==1)*boundElement.z) +  (j == 2)*boundElement.y , // -z if j == 1, y if j == 2
							  (j==0)*boundElement.z  - ((j == 2)*boundElement.x), // z if j == 0, -x if j == 2
							-((j==0)*boundElement.y) +  (j == 1)*boundElement.x , // -y if j == 0, x if j == 1
							0));
						// the second coordinate is the cross product between the normal and the first coordinate
						const float4 coord2 = cross3(boundElement, coord1);

						const float2 vPos0 = vertPos0[neib_index];
						const float2 vPos1 = vertPos1[neib_index];
						const float2 vPos2 = vertPos2[neib_index];

						// relative positions of vertices with respect to the segment, normalized by h
						float4 v0 = (vPos0.x*coord1 + vPos0.y*coord2); // e.g. v0 = r_{v0} - r_s
						float4 v1 = (vPos1.x*coord1 + vPos1.y*coord2);
						float4 v2 = (vPos2.x*coord1 + vPos2.y*coord2);
						float3 vertexWeights;
						const float3 vx[3] = {as_float3(v0), as_float3(v1), as_float3(v2)};
						getMassRepartitionFactor(vx, as_float3(boundElement), vertexWeights);
						float beta = 0.0f;
						if (neibVertXidx == index)
							beta = vertexWeights.x;
						else if (neibVertYidx == index)
							beta = vertexWeights.y;
						else if (neibVertZidx == index)
							beta = vertexWeights.z;

						sumMdot += neibRho*beta*boundElement.w*
									dot3(oldEulerVel[neib_index],boundElement); // the euler vel should be subtracted by the lagrangian vel which is assumed to be 0 now.
					}
				}
			}
			else if (IO_BOUNDARY(info) && FLUID(neib_info)){
				const float4 relPos = pos_corr - oldPos[neib_index];
				const float r = length3(relPos);
				if(!foundFluid && r < influenceradius)
					foundFluid = true;

				// check if this fluid particles is marked for deletion (i.e. vertices != 0)
				if (step == 2 && neibVerts.x | neibVerts.y != 0 && ACTIVE(relPos)) {
					// betaAV is the weight in barycentric coordinates
					float betaAV = 0.0f;
					const float4 vertexWeights = oldGGam[neib_index];
					// check if one of the vertices is equal to the present one
					if (neibVertXidx == index)
						betaAV = vertexWeights.x;
					else if (neibVertYidx == index)
						betaAV = vertexWeights.y;
					else if (neibVertZidx == index)
						betaAV = vertexWeights.z;
					if(betaAV > 0.0f){
						// add mass from fluid particle to vertex particle
						// note that the mass was transfered from pos to gam
						massFluid += betaAV*vertexWeights.w;
					}
				}

			}
		} // BOUNDARY(neib_info) || FLUID(neib_info)
	}

	// normalize wall normal
	if (CORNER(info) || initStep)
		wallNormal = normalize(wallNormal);

	// update boundary conditions on array
	if (!initStep)
		alpha = fmax(alpha, 0.1f*gam); // avoid division by 0
	else
		alpha = fmax(alpha, 1e-5f);
	oldVel[index].w = RHO(sumpWall/alpha,fluid_num(info));
	if (oldTKE)
		oldTKE[index] = fmax(sumtke/numseg, 1e-6f);
	if (oldEps)
		oldEps[index] = fmax(sumeps/numseg, 1e-6f);
	if (!initStep && oldTKE && (!IO_BOUNDARY(info) || CORNER(info) || PRES_IO(info))) {
		// adjust Eulerian velocity so that it is tangential to the fixed wall
		if (CORNER(info))
			// normal for corners is normal to the IO it belongs, so we use wallNormal which is normal
			// to the solid wall it is adjacent to
			as_float3(oldEulerVel[index]) -= dot(as_float3(oldEulerVel[index]), wallNormal)*wallNormal;
		else if (!IO_BOUNDARY(info))
			as_float3(oldEulerVel[index]) -= dot(as_float3(oldEulerVel[index]), normal)*normal;
	}
	// open boundaries
	if (IO_BOUNDARY(info) && !CORNER(info)) {
		float4 eulerVel = oldEulerVel[index];
		if (alpha > 0.1f*oldGGam[index].w) {
			sumvel /= alpha;
			sump /= alpha;
			const float unInt = dot(sumvel, normal);
			const float unExt = dot(as_float3(eulerVel), normal);
			const float rhoInt = RHO(sump, fluid_num(info));
			const float rhoExt = eulerVel.w;

			calculateIOboundaryCondition(eulerVel, info, rhoInt, rhoExt, sumvel, unInt, unExt, normal);
		}
		else {
			if (VEL_IO(info))
				eulerVel.w = d_rho0[fluid_num(info)];
			else
				eulerVel = make_float4(0.0f, 0.0f, 0.0f, eulerVel.w);
		}
		oldEulerVel[index] = eulerVel;
		// the density of the particle is equal to the "eulerian density"
		oldVel[index].w = eulerVel.w;

		// finalize mass computation
		// reference mass:
		const float rho0 = d_rho0[fluid_num(info)];
		const float refMass = deltap*deltap*deltap*rho0;

		// Update vertex mass
		if (!initStep) {
			// time stepping
			pos.w += dt*sumMdot;
			// if a vertex has no fluid particles around and its mass flux is negative then set its mass to 0
			if (alpha < 0.1*gam && sumMdot < 0.0f) // sphynx version
			//if (!foundFluid && sumMdot < 0.0f)
				pos.w = 0.0f;

			// clip to +/- 2 refMass all the time
			pos.w = fmax(-2.0f*refMass, fmin(2.0f*refMass, pos.w));

			// clip to +/- originalVertexMass if we have outflow
			// or if the normal eulerian velocity is less or equal to 0
			if (sumMdot < 0.0f || dot(normal,as_float3(eulerVel)) < 1e-5f*d_sscoeff[fluid_num(info)]) {
				const float4 boundElement = tex1Dfetch(boundTex, index);
				pos.w = fmax(-refMass*boundElement.w, fmin(refMass*boundElement.w, pos.w));
			}

		}
		// particles that have an initial density less than the reference density have their mass set to 0
		// or if their velocity is initially 0
		else if (!resume &&
			( (PRES_IO(info) && eulerVel.w - rho0 <= 1e-10f*rho0) ||
			  (VEL_IO(info) && length3(eulerVel) < 1e-10f*d_sscoeff[fluid_num(info)])) )
			pos.w = 0.0f;

		// check whether new particles need to be created
			// only create new particles in the second part of the time step
		if (step == 2 &&
			// create new particle if the mass of the vertex is large enough
			pos.w > refMass*0.5f &&
			// if mass flux > 0
			sumMdot > 0 &&
			// if imposed velocity is greater 0
			dot(normal,as_float3(eulerVel)) > 1e-5f &&
			// pressure inlets need p > 0 to create particles
			(VEL_IO(info) || eulerVel.w-rho0 > rho0*1e-5f) &&
			// corner vertices are not allowed to create new particles
			!CORNER(info))
		{
			massFluid -= refMass;
			// Create new particle
			particleinfo clone_info;
			uint clone_idx = createNewFluidParticle(clone_info, info, numParticles, numDevices, newNumParticles);

			// Problem has already checked that there is enough memory for new particles
			float4 clone_pos = pos; // new position is position of vertex particle
			clone_pos.w = refMass; // new fluid particle has reference mass
			int3 clone_gridPos = gridPos; // as the position is the same so is the grid position

			// assign new values to array
			oldPos[clone_idx] = clone_pos;
			pinfo[clone_idx] = clone_info;
			particleHash[clone_idx] = calcGridHash(clone_gridPos);
			// the new velocity of the fluid particle is the eulerian velocity of the vertex
			oldVel[clone_idx] = oldEulerVel[index];
			forces[clone_idx] = make_float4(0.0f);

			// the eulerian velocity of fluid particles is always 0
			oldEulerVel[clone_idx] = make_float4(0.0f);
			contupd[clone_idx] = make_float2(0.0f);
			oldGGam[clone_idx] = oldGGam[index];
			vertices[clone_idx] = make_vertexinfo(0, 0, 0, 0);
			if (oldTKE)
				oldTKE[clone_idx] = oldTKE[index];
			if (oldEps)
				oldEps[clone_idx] = oldEps[index];
		}

		// add contribution from newly created fluid or outgoing fluid particles
		pos.w += massFluid;
		oldPos[index].w = pos.w;
	}
	// corners in pressure boundaries have imposed pressures
	//else if (IO_BOUNDARY(info) && CORNER(info) && PRES_IO(info)) {
	//	oldVel[index].w = oldEulerVel[index].w;
	//}

	// finalize computation of average norm for gamma calculation in the initial step
	if (initStep && !resume) {
		oldGGam[index].x = wallNormal.x;
		oldGGam[index].y = wallNormal.y;
		oldGGam[index].z = wallNormal.z;
		oldGGam[index].w = 0.0f;
	}
}

/// Compute the initial value of gamma in the semi-analytical boundary case
/*! This function computes the initial value of \f[\gamma\f] in the semi-analytical boundary case, using a Gauss quadrature formula.
 *	\param[out] newGGam : pointer to the new value of (grad) gamma
 *	\param[in,out] boundelement : normal of segments and of vertices (the latter is computed in this routine)
 *	\param[in] oldPos : pointer to positions and masses; masses of vertex particles are updated
 *	\param[in] oldGGam : pointer to (grad) gamma; used as an approximate normal to the boundary in the computation of gamma
 *	\param[in] vertPos[0] : relative position of the vertex 0 with respect to the segment center
 *	\param[in] vertPos[1] : relative position of the vertex 1 with respect to the segment center
 *	\param[in] vertPos[2] : relative position of the vertex 2 with respect to the segment center
 *	\param[in] pinfo : pointer to particle info; written only when cloning
 *	\param[in] particleHash : pointer to particle hash; written only when cloning
 *	\param[in] cellStart : pointer to indices of first particle in cells
 *	\param[in] neibsList : neighbour list
 *	\param[in] numParticles : number of particles
 *	\param[in] slength : the smoothing length
 *	\param[in] influenceradius : the kernel radius
 */
template<KernelType kerneltype>
__global__ void
__launch_bounds__(BLOCK_SIZE_SHEPARD, MIN_BLOCKS_SHEPARD)
initGamma(
						float4*			newGGam,
						float4*			boundelement,
				const	float4*			oldPos,
				const	float4*			oldGGam,
				const	vertexinfo*		vertices,
				const	uint*			vertIDToIndex,
				const	float2*			vertPos0,
				const	float2*			vertPos1,
				const	float2*			vertPos2,
				const	hashKey*		particleHash,
				const	particleinfo*	pinfo,
				const	uint*			cellStart,
				const	neibdata*		neibsList,
				const	uint			numParticles,
				const	float			slength,
				const	float			deltap,
				const	float			influenceradius,
				const	float			epsilon)
{
	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;

	if (index >= numParticles)
		return;

	// read particle data from sorted arrays
	// kernel is only run for vertex particles
	const particleinfo info = pinfo[index];
	if (BOUNDARY(info))
		return;

	float4 pos = oldPos[index];

	// Compute grid position of current particle
	const int3 gridPos = calcGridPosFromParticleHash( particleHash[index] );

	// Persistent variables across getNeibData calls
	char neib_cellnum = 0;
	uint neib_cell_base_index = 0;
	float3 pos_corr;
	float gam = 1;
	float4 gGam = make_float4(0.f);
	const float3 normal = as_float3(oldGGam[index]);
	float4 newNormal = make_float4(0.0f);

	// Loop over all the neighbors
	for (idx_t i = 0; i < d_neiblist_end; i += d_neiblist_stride) {
		neibdata neib_data = neibsList[i + index];

		if (neib_data == 0xffff) break;

		const uint neib_index = getNeibIndex(pos, pos_corr, cellStart, neib_data, gridPos,
					neib_cellnum, neib_cell_base_index);

		const particleinfo neib_info = pinfo[neib_index];

		if (BOUNDARY(neib_info)) {

			const float4 ns = boundelement[neib_index];
			const float4 relPos = pos_corr - oldPos[neib_index];
			if (INACTIVE(relPos))
				continue;

			// compute new normal for vertices
			if (VERTEX(info)) {
				// prepare ids of neib vertices
				const vertexinfo neibVerts = vertices[neib_index];

				// load the indices of the vertices
				const uint neibVertXidx = vertIDToIndex[neibVerts.x];
				const uint neibVertYidx = vertIDToIndex[neibVerts.y];
				const uint neibVertZidx = vertIDToIndex[neibVerts.z];
				if (index == neibVertXidx || index == neibVertYidx || index == neibVertZidx) {
					if ((IO_BOUNDARY(info) && IO_BOUNDARY(neib_info)) || (!IO_BOUNDARY(info) && !IO_BOUNDARY(neib_info)))
						newNormal += ns;
				}
			}

			// compute gamma for all particles
			// local coordinate system for relative positions to vertices
			uint j = 0;
			// Get index j for which n_s is minimal
			if (fabs(ns.x) > fabs(ns.y))
				j = 1;
			if ((1-j)*fabs(ns.x) + j*fabs(ns.y) > fabs(ns.z))
				j = 2;

			// compute the first coordinate which is a 2-D rotated version of the normal
			const float4 coord1 = normalize(make_float4(
						// switch over j to give: 0 -> (0, z, -y); 1 -> (-z, 0, x); 2 -> (y, -x, 0)
						-((j==1)*ns.z) +  (j == 2)*ns.y , // -z if j == 1, y if j == 2
						(j==0)*ns.z  - ((j == 2)*ns.x), // z if j == 0, -x if j == 2
						-((j==0)*ns.y) +  (j == 1)*ns.x , // -y if j == 0, x if j == 1
						0));
			// the second coordinate is the cross product between the normal and the first coordinate
			const float4 coord2 = cross3(ns, coord1);

			// relative positions of vertices with respect to the segment
			float4 v0 = -(vertPos0[neib_index].x*coord1 + vertPos0[neib_index].y*coord2); // e.g. v0 = r_{v0} - r_s
			float4 v1 = -(vertPos1[neib_index].x*coord1 + vertPos1[neib_index].y*coord2);
			float4 v2 = -(vertPos2[neib_index].x*coord1 + vertPos2[neib_index].y*coord2);
			float4 vertexRelPos[3] = {v0, v1, v2};

			float ggamAS = gradGamma<kerneltype>(slength, as_float3(relPos), vertexRelPos, as_float3(ns));
			float minlRas = 0;
			const float gamAS = Gamma<kerneltype>(slength, as_float3(relPos), vertexRelPos, as_float3(ns), 
					normal, epsilon, deltap, true, minlRas);
			gGam.x += ggamAS*ns.x;
			gGam.y += ggamAS*ns.y;
			gGam.z += ggamAS*ns.z;

			// general formula (also used if particle is on 
			// vertex / edge to compute remaining edges)
			const float x = fmin(dot3(ns, relPos)/slength, 0.25f);
			const float sx = fmax(x*8.0f - 1.0f,0.0f);
			// smootherstep function
			const float smooth = VERTEX(info) ? 1.0f : ((2.0f*sx-5.0f)*3.0f*sx+10.0f)*sx*sx*sx;
			gam -= (smooth > epsilon ? gamAS : 0.0f)*smooth;
		}
	}
	newGGam[index] = make_float4(gGam.x, gGam.y, gGam.z, gam);
	newNormal = normalize3(newNormal);
	boundelement[index] = make_float4(newNormal.x, newNormal.y, newNormal.z, boundelement[index].w);
}

/************************************************************************************************************/
/*					   Kernels for computing acceleration without gradient correction					 */
/************************************************************************************************************/

/* forcesDevice kernel and auxiliary types and functions */
#include "forces_kernel.def"

/************************************************************************************************************/


/************************************************************************************************************/
/*					   Kernels for XSPH, Shepard and MLS corrections									   */
/************************************************************************************************************/

//! This kernel computes the Sheppard correction
template<KernelType kerneltype,
	BoundaryType boundarytype>
__global__ void
__launch_bounds__(BLOCK_SIZE_SHEPARD, MIN_BLOCKS_SHEPARD)
shepardDevice(	const float4*	posArray,
				float4*			newVel,
				const hashKey*		particleHash,
				const uint*		cellStart,
				const neibdata*	neibsList,
				const uint		numParticles,
				const float		slength,
				const float		influenceradius)
{
	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;

	if (index >= numParticles)
		return;

	const particleinfo info = tex1Dfetch(infoTex, index);

	#if( __COMPUTE__ >= 20)
	const float4 pos = posArray[index];
	#else
	const float4 pos = tex1Dfetch(posTex, index);
	#endif

	// If particle is inactive there is absolutely nothing to do
	if (INACTIVE(pos))
		return;

	float4 vel = tex1Dfetch(velTex, index);

	// We apply Shepard normalization :
	//	* with LJ or DYN boundary only on fluid particles
	//TODO 	* with SA boundary ???
	// in any other case we have to copy the vel vector in the new velocity array
	if (NOT_FLUID(info)) {
		newVel[index] = vel;
		return;
	}


	// Taking into account self contribution in summation
	float temp1 = pos.w*W<kerneltype>(0, slength);
	float temp2 = temp1/vel.w ;

	// Compute grid position of current particle
	const int3 gridPos = calcGridPosFromParticleHash( particleHash[index] );

	// Persistent variables across getNeibData calls
	char neib_cellnum = 0;
	uint neib_cell_base_index = 0;
	float3 pos_corr;

	// Loop over all the neighbors
	for (idx_t i = 0; i < d_neiblist_end; i += d_neiblist_stride) {
		neibdata neib_data = neibsList[i + index];

		if (neib_data == 0xffff) break;

		const uint neib_index = getNeibIndex(pos, pos_corr, cellStart, neib_data, gridPos,
					neib_cellnum, neib_cell_base_index);

		// Compute relative position vector and distance
		// Now relPos is a float4 and neib mass is stored in relPos.w
		#if( __COMPUTE__ >= 20)
		const float4 relPos = pos_corr - posArray[neib_index];
		#else
		const float4 relPos = pos_corr - tex1Dfetch(posTex, neib_index);
		#endif


		const particleinfo neib_info = tex1Dfetch(infoTex, neib_index);

		// Skip inactive neighbors
		if (INACTIVE(relPos)) {
			continue;
		}

		const float r = length(as_float3(relPos));

		const float neib_rho = tex1Dfetch(velTex, neib_index).w;

		// Add neib contribution only if it's a fluid one
		// TODO: check with SA
		if ((boundarytype == DYN_BOUNDARY || (boundarytype != DYN_BOUNDARY && FLUID(neib_info)))
				&& r < influenceradius ) {
			const float w = W<kerneltype>(r, slength)*relPos.w;
			temp1 += w;
			temp2 += w/neib_rho;
		}
	}

	// Normalize the density and write in global memory
	vel.w = temp1/temp2;
	newVel[index] = vel;
}

//! This kernel computes the MLS correction
template<KernelType kerneltype,
	BoundaryType boundarytype>
__global__ void
__launch_bounds__(BLOCK_SIZE_MLS, MIN_BLOCKS_MLS)
MlsDevice(	const float4*	posArray,
			float4*			newVel,
			const hashKey*		particleHash,
			const uint*		cellStart,
			const neibdata*	neibsList,
			const uint		numParticles,
			const float		slength,
			const float		influenceradius)
{
	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;

	if (index >= numParticles)
		return;

	const particleinfo info = tex1Dfetch(infoTex, index);

	#if( __COMPUTE__ >= 20)
	const float4 pos = posArray[index];
	#else
	const float4 pos = tex1Dfetch(posTex, index);
	#endif

	// If particle is inactive there is absolutely nothing to do
	if (INACTIVE(pos))
		return;

	float4 vel = tex1Dfetch(velTex, index);

	// We apply MLS normalization :
	//	* with LJ or DYN boundary only on fluid particles
	//TODO 	* with SA boundary ???
	// in any other case we have to copy the vel vector in the new velocity array
	//if (NOT_FLUID(info)) {
	//	newVel[index] = vel;
	//	return;
	//}

	// MLS matrix elements
	symtensor4 mls;
	mls.xx = mls.xy = mls.xz = mls.xw =
		mls.yy = mls.yz = mls.yw =
		mls.zz = mls.zw = mls.ww = 0;

	// Number of neighbors
	int neibs_num = 0;

	// Taking into account self contribution in MLS matrix construction
	mls.xx = W<kerneltype>(0, slength)*pos.w/vel.w;

	// Compute grid position of current particle
	const int3 gridPos = calcGridPosFromParticleHash( particleHash[index] );

	// Persistent variables across getNeibData calls
	char neib_cellnum = 0;
	uint neib_cell_base_index = 0;
	float3 pos_corr;

	// First loop over all neighbors
	for (idx_t i = 0; i < d_neiblist_end; i += d_neiblist_stride) {
		neibdata neib_data = neibsList[i + index];

		if (neib_data == 0xffff) break;

		const uint neib_index = getNeibIndex(pos, pos_corr, cellStart, neib_data, gridPos,
					neib_cellnum, neib_cell_base_index);

		// Compute relative position vector and distance
		// Now relPos is a float4 and neib mass is stored in relPos.w
		#if( __COMPUTE__ >= 20)
		const float4 relPos = pos_corr - posArray[neib_index];
		#else
		const float4 relPos = pos_corr - tex1Dfetch(posTex, neib_index);
		#endif

		// Skip inactive particles
		if (INACTIVE(relPos))
			continue;

		const float r = length(as_float3(relPos));

		const float neib_rho = tex1Dfetch(velTex, neib_index).w;
		const particleinfo neib_info = tex1Dfetch(infoTex, neib_index);

		// Add neib contribution only if it's a fluid one
		// TODO: check with SA
		if (r < influenceradius && (boundarytype == DYN_BOUNDARY || FLUID(neib_info))) {
			neibs_num ++;
			const float w = W<kerneltype>(r, slength)*relPos.w/neib_rho;	// Wij*Vj

			/* Scale relPos by slength for stability and resolution independence */
			MlsMatrixContrib(mls, relPos/slength, w);
		}
	} // end of first loop trough neighbors

	// Resetting persistent variables across getNeibData
	neib_cellnum = 0;
	neib_cell_base_index = 0;

	// We want to compute B solution of M B = E where E =(1, 0, 0, 0) and
	// M is our MLS matrix. M is symmetric, positive (semi)definite. Since we
	// cannot guarantee that the matrix is invertible (it won't be in cases
	// such as thin sheets of particles or structures of even lower topological
	// dimension), we rely on the iterative conjugate residual method to
	// find a solution, with E itself as initial guess.

	// known term
	const float4 E = make_float4(1, 0, 0, 0);

	const float D = det(mls);

	// solution
	float4 B;
	if (fabs(D) < FLT_EPSILON) {
		symtensor4 mls_eps = mls;
		const float eps = fabs(D) + FLT_EPSILON;
		mls_eps.xx += eps;
		mls_eps.yy += eps;
		mls_eps.zz += eps;
		mls_eps.ww += eps;
		const float D_eps = det(mls_eps);
		B = adjugate_row1(mls_eps)/D_eps;
	} else {
		B = adjugate_row1(mls)/D;
	}

#define MAX_CR_STEPS 32
	uint steps = 0;
	for (; steps < MAX_CR_STEPS; ++steps) {
		float lenB = hypot(B);

		float4 MdotB = dot(mls, B);
		float4 residual = E - MdotB;

		// r.M.r
		float num = ddot(mls, residual);

		// (M.r).(M.r)
		float4 Mp = dot(mls, residual);
		float den = dot(Mp, Mp);

		float4 corr = (num/den)*residual;
		float lencorr = hypot(corr);

		if (hypot(residual) < lenB*FLT_EPSILON)
			break;

		if (lencorr < 2*lenB*FLT_EPSILON)
			break;

		B += corr;
	}

	/* Scale for resolution independence, again */
	B.y /= slength;
	B.z /= slength;
	B.w /= slength;

	// Taking into account self contribution in density summation
	vel.w = B.x*W<kerneltype>(0, slength)*pos.w;

	// Loop over all the neighbors (Second loop)
	for (idx_t i = 0; i < d_neiblist_end; i += d_neiblist_stride) {
		neibdata neib_data = neibsList[i + index];

		if (neib_data == 0xffff) break;

		const uint neib_index = getNeibIndex(pos, pos_corr, cellStart, neib_data, gridPos,
			neib_cellnum, neib_cell_base_index);

		// Compute relative position vector and distance
		// Now relPos is a float4 and neib mass is stored in relPos.w
#if( __COMPUTE__ >= 20)
		const float4 relPos = pos_corr - posArray[neib_index];
#else
		const float4 relPos = pos_corr - tex1Dfetch(posTex, neib_index);
#endif

		// Skip inactive particles
		if (INACTIVE(relPos))
			continue;

		const float r = length(as_float3(relPos));

		const particleinfo neib_info = tex1Dfetch(infoTex, neib_index);

		// Interaction between two particles
		if (r < influenceradius && (boundarytype == DYN_BOUNDARY || FLUID(neib_info))) {
			const float w = W<kerneltype>(r, slength)*relPos.w;	 // ρj*Wij*Vj = mj*Wij
			vel.w += MlsCorrContrib(B, relPos, w);
		}
	}  // end of second loop trough neighbors

	// If MLS starts misbehaving, define DEBUG_PARTICLE: this will
	// print the MLS-corrected density for the particles statisfying
	// the DEBUG_PARTICLE condition. Some examples:

//#define DEBUG_PARTICLE (index == numParticles - 1)
//#define DEBUG_PARTICLE (id(info) == numParticles - 1)
//#define DEBUG_PARTICLE (fabs(err) > 64*FLT_EPSILON)

#ifdef DEBUG_PARTICLE
	{
		const float old = tex1Dfetch(velTex, index).w;
		const float err = 1 - vel.w/old;
		if (DEBUG_PARTICLE) {
			printf("MLS %d %d %22.16g => %22.16g (%6.2e)\n",
				index, id(info),
				old, vel.w, err*100);
		}
	}
#endif

	newVel[index] = vel;
}
/************************************************************************************************************/

/************************************************************************************************************/
/*					   CFL max kernel																		*/
/************************************************************************************************************/
//! Computes the max of a float across several threads
template <unsigned int blockSize>
__global__ void
fmaxDevice(float *g_idata, float *g_odata, const uint n)
{
	extern __shared__ float sdata[];

	// perform first level of reduction,
	// reading from global memory, writing to shared memory
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockSize*2 + threadIdx.x;
	unsigned int gridSize = blockSize*2*gridDim.x;

	float myMax = 0;

	// we reduce multiple elements per thread.  The number is determined by the
	// number of active thread blocks (via gridDim).  More blocks will result
	// in a larger gridSize and therefore fewer elements per thread
	while (i < n)
	{
		myMax = max(myMax, g_idata[i]);
		// ensure we don't read out of bounds
		if (i + blockSize < n)
			myMax = max(myMax, g_idata[i + blockSize]);
		i += gridSize;
	}

	// each thread puts its local sum into shared memory
	sdata[tid] = myMax;
	__syncthreads();

	// do reduction in shared mem
	if (blockSize >= 512) { if (tid < 256) { sdata[tid] = myMax = max(myMax,sdata[tid + 256]); } __syncthreads(); }
	if (blockSize >= 256) { if (tid < 128) { sdata[tid] = myMax = max(myMax,sdata[tid + 128]); } __syncthreads(); }
	if (blockSize >= 128) { if (tid <  64) { sdata[tid] = myMax = max(myMax,sdata[tid +  64]); } __syncthreads(); }

	// now that we are using warp-synchronous programming (below)
	// we need to declare our shared memory volatile so that the compiler
	// doesn't reorder stores to it and induce incorrect behavior.
	if (tid < 32)
	{
		volatile float* smem = sdata;
		if (blockSize >=  64) { smem[tid] = myMax = max(myMax, smem[tid + 32]); }
		if (blockSize >=  32) { smem[tid] = myMax = max(myMax, smem[tid + 16]); }
		if (blockSize >=  16) { smem[tid] = myMax = max(myMax, smem[tid +  8]); }
		if (blockSize >=   8) { smem[tid] = myMax = max(myMax, smem[tid +  4]); }
		if (blockSize >=   4) { smem[tid] = myMax = max(myMax, smem[tid +  2]); }
		if (blockSize >=   2) { smem[tid] = myMax = max(myMax, smem[tid +  1]); }
	}

	// write result for this block to global mem
	if (tid == 0)
		g_odata[blockIdx.x] = sdata[0];
}
/************************************************************************************************************/

//! Disables particles that have exited through an open boundary
/*!
 This kernel is only used for SA boundaries in combination with the outgoing particle identification
 in saSegmentBoundaryConditions(). If a particle crosses a segment then the vertexinfo array is set
 for this fluid particle. This is used here to identify such particles. In turn the vertexinfo array
 is reset and the particle is disabled.
*/
__global__ void
disableOutgoingPartsDevice(			float4*		oldPos,
									vertexinfo*	oldVertices,
							const	uint		numParticles)
{
	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;

	if(index < numParticles) {
		const particleinfo info = tex1Dfetch(infoTex, index);
		if (FLUID(info)) {
			float4 pos = oldPos[index];
			if (ACTIVE(pos)) {
				vertexinfo vertices = oldVertices[index];
				if (vertices.x | vertices.y != 0) {
					disable_particle(pos);
					vertices.x = 0;
					vertices.y = 0;
					vertices.z = 0;
					vertices.w = 0;
					oldPos[index] = pos;
					oldVertices[index] = vertices;
				}
			}
		}
	}
}

//! Identify corner vertices on open boundaries
/*!
 Corner vertices are vertices that have segments that are not part of an open boundary. These
 vertices are treated slightly different when imposing the boundary conditions during the
 computation in saVertexBoundaryConditions.
*/
__global__ void
__launch_bounds__(BLOCK_SIZE_SHEPARD, MIN_BLOCKS_SHEPARD)
saIdentifyCornerVertices(
				const	float4*			oldPos,
						particleinfo*	pinfo,
				const	hashKey*		particleHash,
				const	vertexinfo*		vertices,
				const	uint*			cellStart,
				const	neibdata*		neibsList,
				const	uint			numParticles,
				const	float			deltap,
				const	float			eps)
{
	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;

	if (index >= numParticles)
		return;

	// read particle data from sorted arrays
	// kernel is only run for vertex particles which are associated to an open boundary
	particleinfo info = pinfo[index];
	const uint obj = object(info);
	if (!(VERTEX(info) && IO_BOUNDARY(info)))
		return;

	float4 pos = oldPos[index];

	// Compute grid position of current particle
	const int3 gridPos = calcGridPosFromParticleHash( particleHash[index] );

	// Persistent variables across getNeibData calls
	char neib_cellnum = 0;
	uint neib_cell_base_index = 0;
	float3 pos_corr;

	const uint vid = id(info);

	// Loop over all the neighbors
	for (idx_t i = 0; i < d_neiblist_end; i += d_neiblist_stride) {
		neibdata neib_data = neibsList[i + index];

		if (neib_data == 0xffff) break;

		const uint neib_index = getNeibIndex(pos, pos_corr, cellStart, neib_data, gridPos,
					neib_cellnum, neib_cell_base_index);

		const particleinfo neib_info = pinfo[neib_index];
		const uint neib_obj = object(neib_info);

		// loop only over boundary elements that are not of the same open boundary
		if (BOUNDARY(neib_info) && !(obj == neib_obj && IO_BOUNDARY(neib_info))) {
			// check if the current vertex is part of the vertices of the segment
			if (vertices[neib_index].x == vid ||
				vertices[neib_index].y == vid ||
				vertices[neib_index].z == vid) {
				SET_FLAG(info, FG_CORNER);
				pinfo[index] = info;
				break;
			}
		}
	}
}

/** @} */

/************************************************************************************************************/

} //namespace cuforces
#endif
