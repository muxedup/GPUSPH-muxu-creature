#include "hip/hip_runtime.h"
/*  Copyright 2011-2013 Alexis Herault, Giuseppe Bilotta, Robert A. Dalrymple, Eugenio Rustico, Ciro Del Negro

    Istituto Nazionale di Geofisica e Vulcanologia
        Sezione di Catania, Catania, Italy

    Università di Catania, Catania, Italy

    Johns Hopkins University, Baltimore, MD

    This file is part of GPUSPH.

    GPUSPH is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUSPH is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUSPH.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <stdio.h>
#include <stdexcept>

#include "define_buffers.h"
#include "engine_integration.h"
#include "utils.h"

#include "euler_kernel.cu"

#define BLOCK_SIZE_INTEGRATE	256

template<
	SPHFormulation sph_formulation,
	BoundaryType boundarytype,
	KernelType kerneltype,
	flag_t simflags>
class CUDAPredCorrEngine : public AbstractIntegrationEngine
{

void
setconstants(const PhysParams *physparams,
	float3 const& worldOrigin, uint3 const& gridSize, float3 const& cellSize,
	idx_t const& allocatedParticles, int const& maxneibsnum, float const& slength)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_epsxsph), &physparams->epsxsph, sizeof(float)));

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_worldOrigin), &worldOrigin, sizeof(float3)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_cellSize), &cellSize, sizeof(float3)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_gridSize), &gridSize, sizeof(uint3)));
	// Neibs cell to offset table
	char3 cell_to_offset[27];
	for(char z=-1; z<=1; z++) {
		for(char y=-1; y<=1; y++) {
			for(char x=-1; x<=1; x++) {
				int i = (x + 1) + (y + 1)*3 + (z + 1)*9;
				cell_to_offset[i] =  make_char3(x, y, z);
			}
		}
	}
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_cell_to_offset), cell_to_offset, 27*sizeof(char3)));

	idx_t neiblist_end = maxneibsnum*allocatedParticles;
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_neiblist_stride), &allocatedParticles, sizeof(idx_t)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_neiblist_end), &neiblist_end, sizeof(idx_t)));

	const float h3 = slength*slength*slength;
	float kernelcoeff = 1.0f/(M_PI*h3);
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_wcoeff_cubicspline), &kernelcoeff, sizeof(float)));
	kernelcoeff = 15.0f/(16.0f*M_PI*h3);
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_wcoeff_quadratic), &kernelcoeff, sizeof(float)));
	kernelcoeff = 21.0f/(16.0f*M_PI*h3);
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_wcoeff_wendland), &kernelcoeff, sizeof(float)));
}

void
getconstants(PhysParams *physparams)
{
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams->epsxsph, HIP_SYMBOL(cueuler::d_epsxsph), sizeof(float), 0));
}

void
setrbcg(const int3* cgGridPos, const float3* cgPos, int numbodies)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_rbcgGridPos), cgGridPos, numbodies*sizeof(int3)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_rbcgPos), cgPos, numbodies*sizeof(float3)));
}

void
setrbtrans(const float3* trans, int numbodies)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_rbtrans), trans, numbodies*sizeof(float3)));
}

void
setrblinearvel(const float3* linearvel, int numbodies)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_rblinearvel), linearvel, numbodies*sizeof(float3)));
}

void
setrbangularvel(const float3* angularvel, int numbodies)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_rbangularvel), angularvel, numbodies*sizeof(float3)));
}

void
setrbsteprot(const float* rot, int numbodies)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_rbsteprot), rot, 9*numbodies*sizeof(float)));
}

void
basicstep(
		MultiBufferList::const_iterator bufread,
		MultiBufferList::iterator bufreadUpdate,
		MultiBufferList::iterator bufwrite,
		const	uint	*cellStart,
		const	uint	numParticles,
		const	uint	particleRangeEnd,
		const	float	dt,
		const	float	dt2,
		const	int		step,
		const	float	t,
		const	float	slength,
		const	float	influenceradius)
{
	// thread per particle
	uint numThreads = BLOCK_SIZE_INTEGRATE;
	uint numBlocks = div_up(particleRangeEnd, numThreads);

	const float4  *oldPos = bufread->getData<BUFFER_POS>();
	const hashKey *particleHash = bufread->getData<BUFFER_HASH>();
	const float4  *oldVol = bufread->getData<BUFFER_VOLUME>();
	const float4 *oldEulerVel = bufread->getData<BUFFER_EULERVEL>();
	const float *oldTKE = bufread->getData<BUFFER_TKE>();
	const float *oldEps = bufread->getData<BUFFER_EPSILON>();
	const particleinfo *info = bufread->getData<BUFFER_INFO>();
	const neibdata *neibsList = bufread->getData<BUFFER_NEIBSLIST>();
	const float2 * const *vertPos = bufread->getRawPtr<BUFFER_VERTPOS>();

	const float4 *forces = bufread->getData<BUFFER_FORCES>();
	const float2 *contupd = bufread->getData<BUFFER_CONTUPD>();
	const float3 *keps_dkde = bufread->getData<BUFFER_DKDE>();
	const float4 *xsph = bufread->getData<BUFFER_XSPH>();

	// The following two arrays are update in case ENABLE_DENSITY_SUM is set
	// so they are taken from the non-const bufreadUpdate
	float4  *oldVel = bufreadUpdate->getData<BUFFER_VEL>();
	float4 *oldgGam = bufreadUpdate->getData<BUFFER_GRADGAMMA>();

	float4 *newPos = bufwrite->getData<BUFFER_POS>();
	float4 *newVel = bufwrite->getData<BUFFER_VEL>();
	float4 *newVol = bufwrite->getData<BUFFER_VOLUME>();
	float4 *newEulerVel = bufwrite->getData<BUFFER_EULERVEL>();
	float4 *newgGam = bufwrite->getData<BUFFER_GRADGAMMA>();
	float *newTKE = bufwrite->getData<BUFFER_TKE>();
	float *newEps = bufwrite->getData<BUFFER_EPSILON>();
	// boundary elements are updated in-place; only used for rotation in the second step
	float4 *newBoundElement = bufwrite->getData<BUFFER_BOUNDELEMENTS>();

#define ARGS oldPos, particleHash, neibsList, cellStart, oldVel, oldVol, oldEulerVel, oldgGam, oldTKE, oldEps, vertPos,\
	info, forces, contupd, keps_dkde, xsph, newPos, newVel, newVol, newEulerVel, newgGam, newTKE, newEps, newBoundElement, particleRangeEnd, step, dt, dt2, t, slength, influenceradius

	if (step == 1) {
		cueuler::eulerDevice<sph_formulation, boundarytype, kerneltype, simflags><<< numBlocks, numThreads >>>(ARGS);
	} else if (step == 2) {
		cueuler::eulerDevice<sph_formulation, boundarytype, kerneltype, simflags><<< numBlocks, numThreads >>>(ARGS);
	} else {
		throw std::invalid_argument("unsupported predcorr timestep");
	}

#undef ARGS

	// check if kernel invocation generated an error
	CUT_CHECK_ERROR("Euler kernel execution failed");
}

};

