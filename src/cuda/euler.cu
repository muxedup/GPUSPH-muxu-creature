#include "hip/hip_runtime.h"
/*  Copyright 2011-2013 Alexis Herault, Giuseppe Bilotta, Robert A. Dalrymple, Eugenio Rustico, Ciro Del Negro

    Istituto Nazionale di Geofisica e Vulcanologia
        Sezione di Catania, Catania, Italy

    Università di Catania, Catania, Italy

    Johns Hopkins University, Baltimore, MD

    This file is part of GPUSPH.

    GPUSPH is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUSPH is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUSPH.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <stdio.h>
#include <stdexcept>

#include "define_buffers.h"
#include "engine_integration.h"
#include "utils.h"
#include "euler_params.h"

#include "euler_kernel.cu"

#define BLOCK_SIZE_INTEGRATE	256

template<
	SPHFormulation sph_formulation,
	BoundaryType boundarytype,
	KernelType kerneltype,
	ViscosityType visctype,
	flag_t simflags>
class CUDAPredCorrEngine : public AbstractIntegrationEngine
{

void
setconstants(const PhysParams *physparams,
	float3 const& worldOrigin, uint3 const& gridSize, float3 const& cellSize,
	idx_t const& allocatedParticles, int const& maxneibsnum, float const& slength)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_epsxsph), &physparams->epsxsph, sizeof(float)));

	idx_t neiblist_end = maxneibsnum*allocatedParticles;
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_neiblist_stride), &allocatedParticles, sizeof(idx_t)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_neiblist_end), &neiblist_end, sizeof(idx_t)));

	const float h3 = slength*slength*slength;
	float kernelcoeff = 1.0f/(M_PI*h3);
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_wcoeff_cubicspline), &kernelcoeff, sizeof(float)));
	kernelcoeff = 15.0f/(16.0f*M_PI*h3);
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_wcoeff_quadratic), &kernelcoeff, sizeof(float)));
	kernelcoeff = 21.0f/(16.0f*M_PI*h3);
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_wcoeff_wendland), &kernelcoeff, sizeof(float)));
}

void
getconstants(PhysParams *physparams)
{
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams->epsxsph, HIP_SYMBOL(cueuler::d_epsxsph), sizeof(float), 0));
}

void
setrbcg(const int3* cgGridPos, const float3* cgPos, int numbodies)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_rbcgGridPos), cgGridPos, numbodies*sizeof(int3)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_rbcgPos), cgPos, numbodies*sizeof(float3)));
}

void
setrbtrans(const float3* trans, int numbodies)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_rbtrans), trans, numbodies*sizeof(float3)));
}

void
setrblinearvel(const float3* linearvel, int numbodies)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_rblinearvel), linearvel, numbodies*sizeof(float3)));
}

void
setrbangularvel(const float3* angularvel, int numbodies)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_rbangularvel), angularvel, numbodies*sizeof(float3)));
}

void
setrbsteprot(const float* rot, int numbodies)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_rbsteprot), rot, 9*numbodies*sizeof(float)));
}

void
basicstep(
		MultiBufferList::const_iterator bufread,
		MultiBufferList::iterator bufreadUpdate,
		MultiBufferList::iterator bufwrite,
		const	uint	*cellStart,
		const	uint	numParticles,
		const	uint	particleRangeEnd,
		const	float	dt,
		const	float	dt2,
		const	int		step,
		const	float	t,
		const	float	slength,
		const	float	influenceradius)
{
	// thread per particle
	uint numThreads = BLOCK_SIZE_INTEGRATE;
	uint numBlocks = div_up(particleRangeEnd, numThreads);

	const float4  *oldPos = bufread->getData<BUFFER_POS>();
	const hashKey *particleHash = bufread->getData<BUFFER_HASH>();
	const float4  *oldVol = bufread->getData<BUFFER_VOLUME>();
	const float4 *oldEulerVel = bufread->getData<BUFFER_EULERVEL>();
	const float *oldTKE = bufread->getData<BUFFER_TKE>();
	const float *oldEps = bufread->getData<BUFFER_EPSILON>();
	const particleinfo *info = bufread->getData<BUFFER_INFO>();
	const neibdata *neibsList = bufread->getData<BUFFER_NEIBSLIST>();
	const float2 * const *vertPos = bufread->getRawPtr<BUFFER_VERTPOS>();

	const float4 *forces = bufread->getData<BUFFER_FORCES>();
	const float2 *contupd = bufread->getData<BUFFER_CONTUPD>();
	const float3 *keps_dkde = bufread->getData<BUFFER_DKDE>();
	const float4 *xsph = bufread->getData<BUFFER_XSPH>();

	// The following two arrays are update in case ENABLE_DENSITY_SUM is set
	// so they are taken from the non-const bufreadUpdate
	float4  *oldVel = bufreadUpdate->getData<BUFFER_VEL>();
	float4 *oldgGam = bufreadUpdate->getData<BUFFER_GRADGAMMA>();

	float4 *newPos = bufwrite->getData<BUFFER_POS>();
	float4 *newVel = bufwrite->getData<BUFFER_VEL>();
	float4 *newVol = bufwrite->getData<BUFFER_VOLUME>();
	float4 *newEulerVel = bufwrite->getData<BUFFER_EULERVEL>();
	float4 *newgGam = bufwrite->getData<BUFFER_GRADGAMMA>();
	float *newTKE = bufwrite->getData<BUFFER_TKE>();
	float *newEps = bufwrite->getData<BUFFER_EPSILON>();
	// boundary elements are updated in-place; only used for rotation in the second step
	float4 *newBoundElement = bufwrite->getData<BUFFER_BOUNDELEMENTS>();

	euler_params<kerneltype, sph_formulation, boundarytype, visctype, simflags> params(
			newPos, newVel, oldPos, particleHash, oldVel, info, forces, numParticles, dt, dt2, t, step,
			xsph,
			oldgGam, newgGam, contupd, newEulerVel, newBoundElement, vertPos, oldEulerVel, slength, influenceradius, neibsList, cellStart,
			newTKE, newEps, oldTKE, oldEps, keps_dkde,
			newVol, oldVol);

	if (step == 1) {
		cueuler::eulerDevice<kerneltype, sph_formulation, boundarytype, visctype, simflags><<< numBlocks, numThreads >>>(params);
	} else if (step == 2) {
		cueuler::eulerDevice<kerneltype, sph_formulation, boundarytype, visctype, simflags><<< numBlocks, numThreads >>>(params);
	} else {
		throw std::invalid_argument("unsupported predcorr timestep");
	}

#undef ARGS

	// check if kernel invocation generated an error
	KERNEL_CHECK_ERROR;
}

};

