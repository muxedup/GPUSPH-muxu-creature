#include "hip/hip_runtime.h"
/*  Copyright 2011-2013 Alexis Herault, Giuseppe Bilotta, Robert A. Dalrymple, Eugenio Rustico, Ciro Del Negro

    Istituto Nazionale di Geofisica e Vulcanologia
        Sezione di Catania, Catania, Italy

    Università di Catania, Catania, Italy

    Johns Hopkins University, Baltimore, MD

    This file is part of GPUSPH.

    GPUSPH is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUSPH is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUSPH.  If not, see <http://www.gnu.org/licenses/>.
*/

/*
 * Device code.
 */

#ifndef _EULER_KERNEL_
#define _EULER_KERNEL_

#include "particledefine.h"
#include "textures.cuh"
#include "multi_gpu_defines.h"

namespace cueuler {
__constant__ float	d_epsxsph;
__constant__ float3	d_maxlimit;
__constant__ float3	d_minlimit;

__constant__ int3	d_rbcgGridPos[MAX_BODIES]; //< cell of the center of gravity
__constant__ float3	d_rbcgPos[MAX_BODIES]; //< in-cell coordinate of the center of gravity
__constant__ float3	d_rbtrans[MAX_BODIES];
__constant__ float3	d_rblinearvel[MAX_BODIES];
__constant__ float3	d_rbangularvel[MAX_BODIES];
__constant__ float	d_rbsteprot[9*MAX_BODIES];

#include "cellgrid.cuh"

/// Apply rotation to a given vector
/*! Apply the rotation given by the matrix rot to the vector relPos
 * 	should be in the range [0, gridSize.x - 1]x[0, gridSize.y - 1]x[0, gridSize.z - 1].
 *
 *	\param[in] gridHash : hash value
 *
 *	\return grid position
 *
 *	Note : no test is done by this function to ensure that hash value is valid.
 */
__device__ __forceinline__ void
applyrot(const float* rot, const float3 & relPos, float4 & pos)
{
	// Applying rotation
	pos.x += (rot[0] - 1.0f)*relPos.x + rot[1]*relPos.y + rot[2]*relPos.z;
	pos.y += rot[3]*relPos.x + (rot[4] - 1.0f)*relPos.y + rot[5]*relPos.z;
	pos.z += rot[6]*relPos.x + rot[7]*relPos.y + (rot[8] - 1.0f)*relPos.z;
}

__device__ __forceinline__ void
applyrot2(float* rot, float3 & pos, const float3 & cg)
{
	float3 relpos = pos - cg;
	float3 new_relpos;

	// Applying rotation
	new_relpos.x = rot[0]*relpos.x + rot[1]*relpos.y + rot[2]*relpos.z;
	new_relpos.y = rot[3]*relpos.x + rot[4]*relpos.y + rot[5]*relpos.z;
	new_relpos.z = rot[6]*relpos.x + rot[7]*relpos.y + rot[8]*relpos.z;

	pos.x = new_relpos.x + cg.x;
	pos.y = new_relpos.y + cg.y;
	pos.z = new_relpos.z + cg.z;
}


#include "euler_kernel.def"

}
#endif
